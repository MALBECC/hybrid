#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../../../g2g/common.h"
#include "../../../g2g/init.h"
#include "../../../g2g/cuda/cuda_extra.h"
#include "../../../g2g/matrix.h"
#include "../../../g2g/timer.h"
#include "../../../g2g/partition.h"
#include "../../../g2g/scalar_vector_types.h"
#include "../../../g2g/global_memory_pool.h"

//#include "../../../g2g/pointxc/calc_ggaCS.h"
//#include "../../../g2g/pointxc/calc_ggaOS.h"

//#include "../../../g2g/cuda/kernels/accumulate_point.h"

#include "../../../g2g/libxc/libxcproxy.h"
#include "../../../g2g/libxc/libxc_accumulate_point.h"


using namespace std;

using std::cout;
using std::endl;

void accumulate_data_for_libxc_test0001()
{
    printf("** accumulate_data_for_libxc_test0001 **\n");

    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in;

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum;

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *accumulated_density_gpu = NULL;

    // Now the arrays for energy, factors, point_weight and partial_density
    double *energy_gpu_in = NULL;
    double *factor_gpu_in = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);

    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&accumulated_density_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Set the cuda array values to a default value.
    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemset(point_weights_gpu_in, 1, size);
    hipMemset(partial_density_gpu_in, 1, size);
    hipMemset(accumulated_density_gpu, 1, size);

    // Launch the CUDA Kernel
    int numElements = number_of_points;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    // Calculate exc_corr and y2a
    libxc_exchange_correlation_gpu<double, true, true, false> (&libxcProxy,
	energy_gpu_in,
	factor_gpu_in,
	number_of_points,
	accumulated_density_gpu,
	dxyz_gpu_accum.data,
        dd1_gpu_accum.data,
	dd2_gpu_accum.data);

    // Check and print the results.

    // Free memory
    hipFree(energy_gpu_in);
    hipFree(factor_gpu_in);
    hipFree(point_weights_gpu_in);
    hipFree(partial_density_gpu_in);
    hipFree(accumulated_density_gpu);

}


int main()
{
    cout << "Test: Libxc Proxy GPU - BEGIN" << endl;
    accumulate_data_for_libxc_test0001();
    cout << "Test: Libxc Proxy GPU - END" << endl;
    return 0;
}

