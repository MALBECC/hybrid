#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../../../g2g/common.h"
#include "../../../g2g/init.h"
#include "../../../g2g/cuda/cuda_extra.h"
#include "../../../g2g/matrix.h"
#include "../../../g2g/timer.h"
#include "../../../g2g/partition.h"
#include "../../../g2g/scalar_vector_types.h"
#include "../../../g2g/global_memory_pool.h"

//#include "../../../g2g/pointxc/calc_ggaCS.h"
//#include "../../../g2g/pointxc/calc_ggaOS.h"

//#include "../../../g2g/cuda/kernels/accumulate_point.h"

#include "../../../g2g/libxc/libxcproxy.h"
#include "../../../g2g/libxc/libxc_accumulate_point.h"


using namespace std;

using std::cout;
using std::endl;

void accumulate_data_for_libxc_test0001()
{
    printf("** accumulate_data_for_libxc_test0001 **\n");

    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in;

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum;

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *accumulated_density_gpu = NULL;

    // Now the arrays for energy, factors, point_weight and partial_density
    double *energy_gpu_in = NULL;
    double *factor_gpu_in = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);

    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&accumulated_density_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Set the cuda array values to a default value.
    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemset(point_weights_gpu_in, 1, size);
    hipMemset(partial_density_gpu_in, 1, size);
    hipMemset(accumulated_density_gpu, 1, size);

    // Launch the CUDA Kernel
    //int numElements = number_of_points;
    //int threadsPerBlock = 32;
    //int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    //uint block_height = 1;

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    // Calculate exc_corr and y2a
    libxc_exchange_correlation_gpu<double, true, true, false> (&libxcProxy,
	energy_gpu_in,
	factor_gpu_in,
	number_of_points,
	accumulated_density_gpu,
	dxyz_gpu_accum.data,
        dd1_gpu_accum.data,
	dd2_gpu_accum.data);

    // Check and print the results.

    // Free memory
    hipFree(energy_gpu_in);
    hipFree(factor_gpu_in);
    hipFree(point_weights_gpu_in);
    hipFree(partial_density_gpu_in);
    hipFree(accumulated_density_gpu);

}

template <class scalar_type, int width>
__global__ void funcionDeMierda(
		    double* ex, double* exchange,
		    double* ec, double* correlation,
		    double* vrho, double* vrhoC,
		    double* vsigma, double* vsigmaC,
		    double* v2rho, double* v2rhoC,
		    double* v2rhosigma, double* v2rhosigmaC,
		    double* v2sigma, double* v2sigmaC,
		    double* y2a,
		    double* sigma,
		    G2G::vec_type<double, width>* grad,
		    G2G::vec_type<double, width>* hess1,
		    G2G::vec_type<double, width>* hess2,
		    int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        printf("%i %lf %lf %lf\n",i, grad[i].x, grad[i].y, grad[i].z);

	ex[i] = exchange[i];
	ec[i] = correlation[i];
	// Merge the results for the derivatives.
	vrho[i] += vrhoC[i];
        vsigma[i] += vsigmaC[i];
        v2rho[i] += v2rhoC[i];
        v2rhosigma[i] += v2rhosigmaC[i];
        v2sigma[i] += v2sigmaC[i];
        // Now, compute y2a value.
	y2a[i] = vrho[i] - (2 * sigma[i] * v2rhosigma[i]
            + 2 * (hess1[i].x + hess1[i].y + hess1[i].z) * vsigma[i]
            + 4 * v2sigma[i] * (grad[i].x * grad[i].x * hess1[i].x + 
				grad[i].y * grad[i].y * hess1[i].y + 
				grad[i].z * grad[i].z * hess1[i].z + 
				2 * grad[i].x * grad[i].y * hess2[i].x + 
				2 * grad[i].x * grad[i].z * hess2[i].y + 
				2 * grad[i].y * grad[i].z * hess2[i].z));
    }
}


void joinResultsTest0001() {
    printf("joinResultsTest0001()\n");
    // Gather the results.
    // Variables for the Kernels
    int number_of_points = 10;
    int threadsPerBlock = 256;
    int blocksPerGrid = (number_of_points + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t err = hipSuccess;
    int array_size = sizeof(double) * number_of_points;
    
    double* rho = NULL;
    err = hipMalloc((void **)&rho, array_size);
    if (err != hipSuccess) {
	fprintf(stderr, "Failed to allocate device rho! \n");
    }

    double* sigma = NULL;
    err = hipMalloc((void**)&sigma, array_size);
    if (err != hipSuccess) {
	fprintf(stderr, "Failed to allocate device sigma! \n");
    }

    double* exchange = NULL;
    err = hipMalloc((void **)&exchange, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device exchange!\n");
        exit(EXIT_FAILURE);
    }

    double* correlation = NULL;
    err = hipMalloc((void **)&correlation, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device correlation!\n");
        exit(EXIT_FAILURE);
    }
    // Clean arrays
    hipMemset(exchange, 0, array_size);
    hipMemset(correlation, 0, array_size);

    // The outputs for exchange
    double* vrho = NULL;
    double* vsigma = NULL;
    double* v2rho = NULL;
    double* v2rhosigma = NULL;
    double* v2sigma = NULL;

    err = hipMalloc((void **)&vrho, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vrho!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&vsigma, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vsigma!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&v2rho, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device v2rho!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&v2rhosigma, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device v2rhosigma!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&v2sigma, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device v2sigma!\n");
        exit(EXIT_FAILURE);
    }
    // Clear arrays
    hipMemset(vrho, 0, array_size);
    hipMemset(vsigma, 0, array_size);
    hipMemset(v2rho, 0, array_size);
    hipMemset(v2rhosigma, 0, array_size);
    hipMemset(v2sigma, 0, array_size);

    // The outputs for correlation
    double* vrhoC = NULL;
    double* vsigmaC = NULL;
    double* v2rhoC = NULL;
    double* v2rhosigmaC = NULL;
    double* v2sigmaC = NULL;

    err = hipMalloc((void **)&vrhoC, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vrhoC!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&vsigmaC, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vsigmaC!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&v2rhoC, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device v2rhoC!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&v2rhosigmaC, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device v2rhosigmaC!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&v2sigmaC, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device v2sigmaC!\n");
        exit(EXIT_FAILURE);
    }

    double* ex = NULL;
    err = hipMalloc((void **)&ex, array_size);
    if (err != hipSuccess) {
	fprintf(stderr, "Failed to allocate device ex! \n");
    }

    double* ec = NULL;
    err = hipMalloc((void**)&ec, array_size);
    if (err != hipSuccess) {
	fprintf(stderr, "Failed to allocate device ec! \n");
    }

    double* y2a = NULL;
    err = hipMalloc((void **)&y2a, array_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device y2a!\n");
        exit(EXIT_FAILURE);
    }

    // More data
    G2G::CudaMatrix< G2G::vec_type<double,4> > grad;
    G2G::CudaMatrix< G2G::vec_type<double,4> > hess1;
    G2G::CudaMatrix< G2G::vec_type<double,4> > hess2;

    grad.resize(COALESCED_DIMENSION(number_of_points),1);
    hess1.resize(COALESCED_DIMENSION(number_of_points),1);
    hess2.resize(COALESCED_DIMENSION(number_of_points),1);

    grad.zero();
    hess1.zero();
    hess2.zero();

    // Clear arrays
    hipMemset(vrhoC, 0, array_size);
    hipMemset(vsigmaC, 0, array_size);
    hipMemset(v2rhoC, 0, array_size);
    hipMemset(v2rhosigmaC, 0, array_size);
    hipMemset(v2sigmaC, 0, array_size);
    hipMemset(ex, 0, array_size);
    hipMemset(ec, 0, array_size);
    hipMemset(y2a, 0, array_size);

    // Gather the results.
    funcionDeMierda<double, 4><<<blocksPerGrid, threadsPerBlock>>>(
	ex, exchange,
	ec, correlation,
	vrho, vrhoC,
	vsigma, vsigmaC,
	v2rho, v2rhoC,
	v2rhosigma, v2rhosigmaC,
	v2sigma, v2sigmaC,
	y2a,
	sigma,
	grad.data,
	hess1.data,
	hess2.data,
	number_of_points);

    // Print the fucking results
    double* ex_cpu = (double*)malloc(array_size);
    double* ec_cpu = (double*)malloc(array_size);
    double* y2a_cpu = (double*)malloc(array_size);

    hipMemcpy(ex_cpu, ex, array_size, hipMemcpyDeviceToHost);
    hipMemcpy(ec_cpu, ec, array_size, hipMemcpyDeviceToHost);
    hipMemcpy(y2a_cpu, y2a, array_size, hipMemcpyDeviceToHost);

    printf("Results \n");
    for (int j=0; j<number_of_points; j++) {
	printf("%i %lf %lf %lf \n",j, ex_cpu[j], ec_cpu[j], y2a_cpu[j]);
    }

    // Free device memory.
    if (rho != NULL) {
	hipFree(rho);
    }
    if (sigma != NULL) {
	hipFree(sigma);
    }
    if (exchange != NULL) {
	hipFree(exchange);
    }
    if (correlation != NULL) {
	hipFree(correlation);
    }
    if (vrho != NULL) {
        hipFree(vrho);
    }
    if (vsigma != NULL) {
	hipFree(vsigma);
    }
    if (v2rho != NULL) {
	hipFree(v2rho);
    }
    if (v2rhosigma != NULL) {
	hipFree(v2rhosigma);
    }
    if (v2sigma != NULL) {
	hipFree(v2sigma);
    }
    if (vrhoC != NULL) {
        hipFree(vrhoC);
    }
    if (vsigmaC != NULL) {
	hipFree(vsigmaC);
    }
    if (v2rhoC != NULL) {
	hipFree(v2rhoC);
    }
    if (v2rhosigmaC != NULL) {
	hipFree(v2rhosigmaC);
    }
    if (v2sigmaC != NULL) {
	hipFree(v2sigmaC);
    }

    free(ex_cpu);
    free(ec_cpu);
    free(y2a_cpu);
}

int main()
{
    cout << "Test: Libxc Proxy GPU - BEGIN" << endl;
    //accumulate_data_for_libxc_test0001();
    joinResultsTest0001();
    cout << "Test: Libxc Proxy GPU - END" << endl;
    return 0;
}

