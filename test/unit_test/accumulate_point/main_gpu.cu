#include "hip/hip_runtime.h"
//#ifndef GPU_KENERLS
//#define GPU_KERNELS
//#endif

#include <cassert>
#include <fstream>
#include <iostream>
#include <limits>
#include <map>
#include <hip/hip_math_constants.h>
#include <float.h>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include "../../../g2g/common.h"
#include "../../../g2g/init.h"
#include "../../../g2g/cuda/cuda_extra.h"
#include "../../../g2g/matrix.h"
#include "../../../g2g/timer.h"
#include "../../../g2g/partition.h"
#include "../../../g2g/scalar_vector_types.h"
#include "../../../g2g/global_memory_pool.h"

//#include "../../../g2g/pointxc/calc_ggaCS.h"
//#include "../../../g2g/pointxc/calc_ggaOS.h"

//#include "../../../g2g/cuda/kernels/accumulate_point.h"

#include "../../../g2g/libxc/libxcproxy.h"
#include "../../../g2g/libxc/libxc_accumulate_point.h"

//////////////////////////////////////
//// CALC_GGA
#define POT_ALPHA     ((scalar_type)-0.738558766382022447)
#define POT_GL        ((scalar_type)0.620350490899400087)

#define POT_VOSKO_A1  ((scalar_type)0.03109205)
#define POT_VOSKO_B1  ((scalar_type)3.72744)
#define POT_VOSKO_C1  ((scalar_type)12.9352)
#define POT_VOSKO_X0  ((scalar_type)-0.10498)
#define POT_VOSKO_Q   ((scalar_type)6.15199066246304849)
#define POT_VOSKO_A16 ((scalar_type)0.005182008333)
#define POT_VOSKO_Q2  ((scalar_type)4.7309269)

#define POT_ALYP  ((scalar_type)0.04918)
#define POT_BLYP  ((scalar_type)0.132)
#define POT_CLYP  ((scalar_type)0.2533)
#define POT_CLYP3 ((scalar_type)0.0844333333)
#define POT_DLYP  ((scalar_type)0.349)
#define POT_DLYP3 ((scalar_type)0.116333333)
#define POT_CF    ((scalar_type)2.87123400018819)
#define POT_BETA  ((scalar_type)0.0042)

#define POT_ALF ((scalar_type)0.023266)
#define POT_BET ((scalar_type)7.389)
#define POT_GAM ((scalar_type)8.723)
#define POT_DEL ((scalar_type)0.472)

//////////////////////////////////////
//// UTILS

void set_array_values (double* data, double value,  int size) 
{
    if (data == NULL)
    {
	return;
    } else {
	for (int i=0; i<size; i++) {
	    data[i] = value;
	}
    }
}

//////////////////////////////////////
//// PRINT UTILS

void print_array (double* data, int size) 
{
    printf ("[");
    if (data == NULL) {
	printf("empty");
    } else {
	for (int i=0; i<size; i++) {
	    printf("%e,", data[i]);
	}
    }
    printf("]\n");
}

void print_vec_type (G2G::vec_type<double,4>* data, int size) 
{
    printf ("[");
    if (data == NULL) {
	printf("empty");
    } else {
	for (int i=0; i<size; i++) {
	    printf("(%f,%f,%f),", data[i].x, data[i].y, data[i].z);
	}
    }
    printf("]\n");
}


void print_accumulate_point_data (G2G::vec_type<double,4>* dxyz, G2G::vec_type<double,4>* dd1, 
    G2G::vec_type<double,4>* dd2, double* energy, double* factor, double* point_weights,
    double* partial_density, int number_of_points)
{
    printf("=========\n");
    printf("= Data  =\n");
    printf("=========\n");
    printf("dxyz:"); print_vec_type(dxyz, number_of_points);
    printf("dd1:"); print_vec_type(dd1, number_of_points);
    printf("dd2:"); print_vec_type(dd2, number_of_points);
    printf("energy:"); print_array(energy, number_of_points);
    printf("factor:"); print_array(factor, number_of_points);
    printf("point_weights:"); print_array(point_weights, number_of_points);
    printf("partial_density:"); print_array(partial_density, number_of_points);
    printf("=========\n");

}


//////////////////////////////////////
//// CALC_GGACS

template<class scalar_type, int iexch, unsigned int width>  __device__
void calc_ggaCS( scalar_type dens, 
                 const G2G::vec_type<scalar_type,width>& grad,
                 const G2G::vec_type<scalar_type,width>& hess1,
                 const G2G::vec_type<scalar_type,width>& hess2,
                 scalar_type& ex, 
                 scalar_type& ec, 
                 scalar_type& y2a)
{
   // hess1: xx, yy, zz  || hess2: xy, xz, yz
   const scalar_type MINIMUM_DENSITY_VALUE = 1e-13f;
   if (dens < MINIMUM_DENSITY_VALUE) { ex = ec = 0; return; }

   scalar_type y     = cbrt( (scalar_type)dens );  // rho^(1/3)
   scalar_type grad2 = grad.x * grad.x + grad.y * grad.y + grad.z * grad.z;
   if (grad2 == 0) grad2 = (scalar_type)FLT_MIN;
   scalar_type dgrad = sqrt(grad2);

   scalar_type d0 = hess1.x + hess1.y + hess1.z;
   scalar_type u0 = ((grad.x * grad.x) * hess1.x 
                  + 2.0 * grad.x * grad.y * hess2.x 
                  + 2.0 * grad.y * grad.z * hess2.z 
                  + 2.0 * grad.x * grad.z * hess2.y 
                  + (grad.y * grad.y) * hess1.y 
                  + (grad.z * grad.z) * hess1.z) / dgrad;
   y2a = 0;

   // Exchange - Perdew : Phys. Rev B 33 8800 (1986)
   if (iexch == 4 || iexch == 8) {
      scalar_type dens2 = (dens * dens);
      scalar_type ckf   = (scalar_type)3.0936677 * y;
      scalar_type s     = dgrad / ((scalar_type)2.0 * ckf * dens);

      scalar_type fx = (1.0 / 15.0);
      scalar_type s2 = (s * s);
      scalar_type s3 = (s * s * s);
      scalar_type g0 = 1.0 + 1.296 * s2 + 14.0 * pow(s, 4) + 0.2 * pow(s, 6);
      scalar_type F  = pow(g0, fx);
      scalar_type e  = POT_ALPHA * F * y;
      ex = e;

      scalar_type t = d0 / (dens * 4.0 * (ckf * ckf));
      scalar_type u = u0 / (pow( (scalar_type)2.0 * ckf, 3) * dens2);

      scalar_type g2  = 2.592 * s + 56.0 * s3 + 1.2 * pow(s, 5);
      scalar_type g3  = 2.592     + 56.0 * s2 + 1.2 * pow(s, 4);
      scalar_type g4  = 112.0 * s + 4.8  * s3;
      scalar_type dF  = fx * F/g0 * g2;
      scalar_type dsF = fx * F/g0 * (-14.0 * fx * g3 * g2/g0 + g4);

      y2a = POT_ALPHA * y * (1.33333333333 * F - t/s * dF - (u-1.3333333333 * s3) * dsF);
   } else if (iexch >= 5 && iexch <= 7) { // Becke  : Phys. Rev A 38 3098 (1988)
      scalar_type e0 = POT_ALPHA * y;
      scalar_type y2 = dens / 2.0;
      scalar_type r13 = cbrt( y2 );
      scalar_type r43 = cbrt( pow(y2, 4) );
      scalar_type Xs = dgrad / (2.0 * r43);
      scalar_type siper = asinh(Xs);
      scalar_type DN = 1.0 + 6.0 * POT_BETA * Xs * siper;
      scalar_type ect = -2.0 * POT_BETA * r43 * Xs * Xs/(DN * dens);
      scalar_type e = e0 + ect;
      ex = e;

      // Potential
      scalar_type v0 = 1.33333333333333 * e0;
      scalar_type Fb = 1.0 / DN;
      scalar_type XA1 = Xs / sqrt(1.0 + Xs * Xs);
      scalar_type DN1 = 1.0 + Fb * (1.0 - 6.0 * POT_BETA * Xs * XA1);
      scalar_type DN2 = 1.0 / (1.0 + Xs * Xs) + 2.0 * Fb * (2.0 - 6.0 * POT_BETA * Xs * XA1);
      scalar_type DN3 = siper * (1.0 + 2.0 * Fb) + XA1 * DN2;
      scalar_type D02 = d0 / 2.0;
      scalar_type de1 = 1.33333333333333 / (cbrt(pow((scalar_type)dens,7) ) );

      scalar_type DGRADx = (grad.x * hess1.x + grad.y * hess2.x + grad.z * hess2.y) / dgrad;
      scalar_type GRADXx = cbrt( (scalar_type) 2.0 ) * (1.0 / (dens * y) * DGRADx - de1 * grad.x * dgrad);
      scalar_type DGRADy = (grad.x * hess2.x + grad.y * hess1.y + grad.z * hess2.z) / dgrad;
      scalar_type GRADXy = cbrt( (scalar_type) 2.0 ) * (1.0 / (dens * y) * DGRADy - de1 * grad.y * dgrad);
      scalar_type DGRADz = (grad.x * hess2.y + grad.y * hess2.z + grad.z * hess1.z) / dgrad;
      scalar_type GRADXz = cbrt( (scalar_type) 2.0 ) * (1.0 / (dens * y) * DGRADz - de1 * grad.z * dgrad);

      scalar_type T1   = grad.x / 2.0 * GRADXx;
      scalar_type T2   = grad.y / 2.0 * GRADXy;
      scalar_type T3   = grad.z / 2.0 * GRADXz;
      scalar_type DN4  = 6.0 * POT_BETA * Fb * (T1 + T2 + T3);
      scalar_type DN5  = 1.33333333333333 * r43 * r13 * Xs * Xs;
      scalar_type TOT2 = DN5 - D02 * DN1 + DN4 * DN3;

      scalar_type vxc = -POT_BETA * Fb/r43 * TOT2;
      y2a = v0 + vxc;
   } else { // PBE
      //scalar_type dgrad2 = grad.y * grad.y * hess1.y;
      //scalar_type dgrad3 = grad.z * grad.z * hess1.z;
      //scalar_type dgrad4 = grad.x * grad.y * hess2.x;
      //scalar_type dgrad5 = grad.x * grad.z * hess2.y;
      //scalar_type dgrad6 = grad.y * grad.z * hess2.z;
      //scalar_type delgrad = (dgrad1 + dgrad2 + dgrad3 + 2 * (dgrad4 + dgrad5 + dgrad6)) / dgrad;
      //scalar_type rlap = hess1.x + hess1.y + hess1.z;

      //scalar_type expbe, vxpbe, ecpbe, vcpbe;
      //pbeCS(dens, dgrad, delgrad, rlap, expbe, vxpbe, ecpbe, vcpbe);

      //ex  = expbe;
      //ec  = ecpbe;
      //y2a = vxpbe + vcpbe;
      return;
   }

   // Correlation - Perdew : Phys. Rev B 33 8822 (1986)
   if (iexch >= 4 && iexch <= 6) {
      // TO-DO: hay algun problema con 4 y 5, probablemente este aca
      scalar_type dens2 = (dens * dens);
      scalar_type rs  = POT_GL / y;
      scalar_type x1  = sqrt(rs);
      scalar_type Xx  = rs + POT_VOSKO_B1 * x1 + POT_VOSKO_C1;
      scalar_type Xxo = (POT_VOSKO_X0 * POT_VOSKO_X0) 
                      + POT_VOSKO_B1 * POT_VOSKO_X0 + POT_VOSKO_C1;
  
      scalar_type t1 = 2.0 * x1 + POT_VOSKO_B1;
      scalar_type t2 = log(Xx);
      scalar_type t3 = atan(POT_VOSKO_Q/t1);
      scalar_type t4 = POT_VOSKO_B1 * POT_VOSKO_X0/Xxo;

      ec = POT_VOSKO_A1 * ( 2.0 * log(x1) - t2 
           + 2.0 * POT_VOSKO_B1/POT_VOSKO_Q * t3
           - t4 *(2.0 * log(x1 - POT_VOSKO_X0) - t2 
           + 2.0 * (POT_VOSKO_B1 + 2.0 * POT_VOSKO_X0) / POT_VOSKO_Q * t3));

      scalar_type t5 = (POT_VOSKO_B1 * x1 + 2.0 * POT_VOSKO_C1) / x1;
      scalar_type t6 = POT_VOSKO_X0 / Xxo;
      scalar_type vc = ec - POT_VOSKO_A16 * x1 * 
                   ( t5/Xx - 4.0 * POT_VOSKO_B1 / ((t1 * t1)+(POT_VOSKO_Q * POT_VOSKO_Q2)) 
                   * (1.0 - t6 * (POT_VOSKO_B1 - 2.0 * POT_VOSKO_X0)) 
                   - t4 * (2.0 / (x1 - POT_VOSKO_X0) - t1/Xx));

      if (iexch == 6) {
         y2a = y2a + vc;
      } else {
         scalar_type rs2 = (rs * rs);
         scalar_type Cx1 = 0.002568f + POT_ALF * rs + POT_BET * rs2;
         scalar_type Cx2 = 1.0f + POT_GAM * rs + POT_DEL * rs2 + 1.0e4 * POT_BET * (rs * rs * rs);
         scalar_type C   = 0.001667 + Cx1/Cx2;
         scalar_type Cx3 = POT_ALF + 2.0f * POT_BET * rs;
         scalar_type Cx4 = POT_GAM + 2.0f * POT_DEL * rs + 3.0e4 * POT_BET * rs2;
         scalar_type dC  = Cx3/Cx2 - Cx1/(Cx2 * Cx2) * Cx4;
         dC = -0.333333333333333f * dC * POT_GL / (y * dens);

         scalar_type phi  = 0.0008129082f/C * dgrad/pow((scalar_type)dens, (scalar_type)(7.0f/6.0f));
         scalar_type expo = exp(-phi);
         scalar_type ex0  = expo * C;

         ec = ec + ex0 * grad2 / (y * dens2);

         scalar_type D1   = (2.0f - phi) * d0/dens;
         scalar_type phi2 = (phi * phi);
         scalar_type D2   = 1.33333333333333333f - 3.666666666666666666f * phi + 1.166666666666666f * phi2;
         D2 = D2 * grad2/dens2;
         scalar_type D3 = phi * (phi - 3.0f) * u0/(dens * dgrad);
         scalar_type D4 = expo * grad2 / (y * dens) * (phi2 - phi - 1.0f) * dC;

         vc = vc - 1.0 * (ex0 / y * (D1 - D2 + D3) - D4);
         y2a = y2a + vc;
      }
   } else if (iexch == 7 || iexch == 8) { // Correlation - LYP: PRB 37 785 (1988)
      scalar_type rom13 = 1 / cbrt( dens );
      scalar_type rom53 = cbrt( pow(dens, 5) );
      scalar_type ecro  = expf(-POT_CLYP * rom13);
      scalar_type f1    = 1.0f / (1.0f + POT_DLYP * rom13);
      scalar_type tw    = 1.0f / 8.0f * (grad2/dens - d0);
      scalar_type term  = (tw / 9.0f + d0 / 18.0f) - 2.0f * tw + POT_CF * rom53;
      term = dens + POT_BLYP * (rom13 * rom13) * ecro * term;

      ec = -POT_ALYP * f1 * term/dens;

      scalar_type h1 = ecro/rom53;
      scalar_type g1 = f1 * h1;
      scalar_type tm1 = POT_DLYP3 * (rom13/dens);
      scalar_type fp1 = tm1 * (f1 * f1);
      scalar_type tm2 = -1.666666666f + POT_CLYP3 * rom13;
      scalar_type hp1 = h1 * tm2/dens;
      scalar_type gp1 = fp1 * h1 + hp1 * f1;
      scalar_type fp2 = tm1 * 2.0f * f1 * (fp1 - 0.6666666666f * f1/dens);
      scalar_type tm3 = 1.6666666666f - POT_CLYP3 * 1.3333333333f * rom13;
      scalar_type hp2 = hp1 * tm2/dens + h1 * tm3/(dens * dens);
      scalar_type gp2 = fp2 * h1 + 2.0f * fp1 * hp1 + hp2 * f1;

      scalar_type term3 = -POT_ALYP * (fp1 * dens + f1) 
                          -POT_ALYP * POT_BLYP * POT_CF * (gp1 * dens + 8.0f/3.0f * g1) * rom53;
      scalar_type term4 = (gp2 * dens * grad2 + gp1 * (3.0f * grad2 + 2.0f * dens * d0)
                          + 4.0f * g1 * d0) * POT_ALYP * POT_BLYP/4.0f;
      scalar_type term5 = (3.0f * gp2 * dens * grad2 + gp1 * (5.0f * grad2 + 6.0f * dens * d0)
                          + 4.0f * g1 * d0) * POT_ALYP * POT_BLYP/72.0f;

      y2a = y2a + (term3 - term4 - term5);
   }
}

template<class scalar_type, unsigned int width> __device__
void calc_ggaCS_in( scalar_type dens, 
                    const G2G::vec_type<scalar_type,width>& grad,
                    const G2G::vec_type<scalar_type,width>& hess1,
                    const G2G::vec_type<scalar_type,width>& hess2,
                    scalar_type& ex, 
                    scalar_type& ec, 
                    scalar_type& y2a,
                    const int iexch)
{
   switch(iexch) {
      case 0: return calc_ggaCS<scalar_type, 0, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      case 1: return calc_ggaCS<scalar_type, 1, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      case 2: return calc_ggaCS<scalar_type, 2, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      case 3: return calc_ggaCS<scalar_type, 3, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      case 4: return calc_ggaCS<scalar_type, 4, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      case 5: return calc_ggaCS<scalar_type, 5, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      case 6: return calc_ggaCS<scalar_type, 6, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      case 7: return calc_ggaCS<scalar_type, 7, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      case 8: return calc_ggaCS<scalar_type, 8, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      case 9: return calc_ggaCS<scalar_type, 9, width>(dens, grad, hess1, hess2, ex, ec, y2a);
      default: assert(false);
   }
}



//////////////////////////////////////
//// KERNEL FOR ACCUMULATE_POINT

template<class scalar_type, bool compute_energy, bool compute_factor, bool lda>
__global__ void gpu_accumulate_point (scalar_type* const energy, scalar_type* const factor, 
		    const scalar_type* const point_weights,
            	    uint points, int block_height, scalar_type* partial_density, 
		    G2G::vec_type<scalar_type,4>* dxyz,
                    G2G::vec_type<scalar_type,4>* dd1, 
		    G2G::vec_type<scalar_type,4>* dd2) {

  uint point = blockIdx.x * DENSITY_ACCUM_BLOCK_SIZE + threadIdx.x;
  //uint point = blockIdx.x * 128 + threadIdx.x;

  scalar_type point_weight = 0.0f;
  scalar_type y2a, exc_corr, exc_c, exc_x;

  scalar_type _partial_density(0.0f);
  G2G::vec_type<scalar_type,4> _dxyz, _dd1, _dd2;

  _dxyz = _dd1 = _dd2 = G2G::vec_type<scalar_type,4>(0.0f,0.0f,0.0f,0.0f);

  bool valid_thread = (point < points);
  if (valid_thread)
    point_weight = point_weights[point];

  if (valid_thread) {
    for(int j =0 ; j<block_height; j++) {
      const int this_row = j*points+point;

      _partial_density += partial_density[this_row];
      _dxyz += dxyz[this_row];
      _dd1 += dd1[this_row];
      _dd2 += dd2[this_row];
     }
  }

  calc_ggaCS_in<scalar_type, 4>(_partial_density, _dxyz, _dd1, _dd2, exc_x, exc_c, y2a, 9);
  exc_corr = exc_x + exc_c;

  if (compute_energy && valid_thread){
    energy[point] = (_partial_density * point_weight) * exc_corr;
  }

  if (compute_factor && valid_thread){
    factor[point] = point_weight * y2a;
  }

}


//////////////////////////////////////
//// TESTS

void gpu_accumulate_point_test0001()
{
    printf("** gpu_accumulate_point_test0001 **\n");

    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu;

    dxyz_gpu.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *energy_gpu = NULL;
    double *factor_gpu = NULL;
    double *point_weights_gpu = NULL;
    double *partial_density_gpu = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);
    err = hipMalloc((void**)&energy_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    err = hipMalloc((void**)&point_weights_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu!\n");
    }

    // Set the cuda array values to a default value.
    hipMemset(energy_gpu, 0, size);
    hipMemset(factor_gpu, 0, size);
    hipMemset(point_weights_gpu, 1, size);
    hipMemset(partial_density_gpu, 1, size);

    // Launch the CUDA Kernel
    int numElements = n+m;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    // Call the CUDA KERNEL
    gpu_accumulate_point<double,true, true, false><<<blocksPerGrid, threadsPerBlock>>> 
		    (energy_gpu, factor_gpu, 
		    point_weights_gpu,
            	    numElements, 1, partial_density_gpu, 
		    dxyz_gpu.data,
                    dd1_gpu.data, 
		    dd2_gpu.data);


    // Allocate the host input vectors
    double *energy_cpu = (double *)malloc(size);
    double *factor_cpu = (double *)malloc(size);
    double *point_weights_cpu = (double *)malloc(size);
    double *partial_density_cpu = (double *)malloc(size);

    // Copy the vectors from gpu to cpu
    // Be aware that energy_gpu can be NULL.
    err = hipMemcpy(energy_cpu, energy_gpu, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector energy_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(factor_cpu, factor_gpu, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector factor_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(point_weights_cpu, point_weights_gpu, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector point_weights_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(partial_density_cpu, partial_density_gpu, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector partial_density_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    // Before print the results, get the data from cuda
    G2G::vec_type<double,4>* dxyz_cpu;
    G2G::vec_type<double,4>* dd1_cpu;
    G2G::vec_type<double,4>* dd2_cpu;

    // Alloc memory in the host for the gpu data
    uint cpu_vecs_size = number_of_points * sizeof(G2G::vec_type<double,4>);
    dxyz_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);
    dd1_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);
    dd2_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);

    // Copy data from device to host.
    err = hipMemcpy(dxyz_cpu, dxyz_gpu.data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dxyz_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dd1_cpu, dd1_gpu.data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dd1_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dd2_cpu, dd2_gpu.data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dd2_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (dxyz_cpu, dd1_cpu, dd2_cpu, energy_cpu, 
	factor_cpu, point_weights_cpu, 
	partial_density_cpu, number_of_points);

    free(energy_cpu);
    free(factor_cpu);
    free(point_weights_cpu);
    free(partial_density_cpu);

    free(dxyz_cpu);
    free(dd1_cpu);
    free(dd2_cpu);

    hipFree(energy_gpu);
    hipFree(factor_gpu);
    hipFree(point_weights_gpu);
    hipFree(partial_density_gpu);

}

void accumulate_data_for_libxc_test0001()
{
    printf("** accumulate_data_for_libxc_test0001 **\n");

    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in;

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum;

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *partial_density_gpu_accum = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);
    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_accum, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Set the cuda array values to a default value.
    hipMemset(point_weights_gpu_in, 1, size);
    hipMemset(partial_density_gpu_in, 1, size);
    hipMemset(partial_density_gpu_accum, 1, size);

    // Launch the CUDA Kernel
    int numElements = n+m;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // Call the CUDA KERNEL
    gpu_accumulate_point_for_libxc<double,true,true,false><<<blocksPerGrid,threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in.data, dd1_gpu_in.data, dd2_gpu_in.data,
	partial_density_gpu_accum, dxyz_gpu_accum.data, dd1_gpu_accum.data, dd2_gpu_accum.data);

    // Free memory
    hipFree (point_weights_gpu_in);
    hipFree (partial_density_gpu_in);
    hipFree (partial_density_gpu_accum);

}


void accumulate_data_for_libxc_test0002()
{
    printf("** accumulate_data_for_libxc_test0002 **\n");

    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in;

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum;

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *partial_density_gpu_accum = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);
    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_accum, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Set the cuda array values to a default value.
    hipMemset(point_weights_gpu_in, 1, size);
    hipMemset(partial_density_gpu_in, 1, size);
    hipMemset(partial_density_gpu_accum, 1, size);

    // Launch the CUDA Kernel
    int numElements = n+m;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // Call the CUDA KERNEL
    gpu_accumulate_point_for_libxc<double,true,true,false><<<blocksPerGrid,threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in.data, dd1_gpu_in.data, dd2_gpu_in.data,
	partial_density_gpu_accum, dxyz_gpu_accum.data, dd1_gpu_accum.data, dd2_gpu_accum.data);

    // With the accumulate parameters we can use
    // the data for libxc.
    G2G::vec_type<double,4>* dxyz_cpu;
    G2G::vec_type<double,4>* dd1_cpu;
    G2G::vec_type<double,4>* dd2_cpu;

    // Alloc memory in the host for the gpu data
    uint cpu_vecs_size = number_of_points * sizeof(G2G::vec_type<double,4>);
    dxyz_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);
    dd1_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);
    dd2_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);

    // Copy data from device to host.
    err = hipMemcpy(dxyz_cpu, dxyz_gpu_accum.data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dxyz_gpu_accum from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dd1_cpu, dd1_gpu_accum.data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dd1_gpu_accum from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dd2_cpu, dd2_gpu_accum.data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dd2_gpu_accum from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (dxyz_cpu, dd1_cpu, dd2_cpu, NULL, 
	NULL, NULL, NULL, number_of_points);

    // Free memory
    free(dxyz_cpu);
    free(dd1_cpu);
    free(dd2_cpu);

    hipFree (point_weights_gpu_in);
    hipFree (partial_density_gpu_in);
    hipFree (partial_density_gpu_accum);

}


void accumulate_data_for_libxc_test0003()
{
    printf("** accumulate_data_for_libxc_test0003 **\n");

    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in;

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum;

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *accumulated_density_gpu = NULL;

    // Now the arrays for energy, factors, point_weight and partial_density
    double *energy_gpu_in = NULL;
    double *factor_gpu_in = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);

    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&accumulated_density_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Set the cuda array values to a default value.
    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemset(point_weights_gpu_in, 1, size);
    hipMemset(partial_density_gpu_in, 1, size);
    hipMemset(accumulated_density_gpu, 1, size);

    // Launch the CUDA Kernel
    int numElements = number_of_points;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // Call the CUDA KERNEL
    gpu_accumulate_point_for_libxc<double,true,true,false><<<blocksPerGrid,threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in.data, dd1_gpu_in.data, dd2_gpu_in.data,
	accumulated_density_gpu, dxyz_gpu_accum.data, dd1_gpu_accum.data, dd2_gpu_accum.data);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 101;
    const int functionalCorrelation = 130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    // Calculate exc_corr and y2a
    libxc_exchange_correlation_cpu<double, true, true, false> (&libxcProxy,
	energy_gpu_in,
	factor_gpu_in,
	number_of_points,
	accumulated_density_gpu,
	dxyz_gpu_accum.data,
        dd1_gpu_accum.data,
	dd2_gpu_accum.data);

    // TODO: queda unir los resultados.
    // con la funcion gpu_accumulate_energy_and_factor_from_libxc(...);

    // Free memory
    hipFree(energy_gpu_in);
    hipFree(factor_gpu_in);
    hipFree(point_weights_gpu_in);
    hipFree(partial_density_gpu_in);
    hipFree(accumulated_density_gpu);

}


void accumulate_data_for_libxc_test0004()
{
    printf("** accumulate_data_for_libxc_test0004 **\n");

    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in;

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum;

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *accumulated_density_gpu = NULL;

    // Now the arrays for energy, factors, point_weight and partial_density
    double *energy_gpu_in = NULL;
    double *factor_gpu_in = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);

    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&accumulated_density_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Set the cuda array values to a default value.
    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemset(point_weights_gpu_in, 1, size);
    hipMemset(partial_density_gpu_in, 1, size);
    hipMemset(accumulated_density_gpu, 1, size);

    // Launch the CUDA Kernel
    int numElements = number_of_points;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // Call the CUDA KERNEL
    gpu_accumulate_point_for_libxc<double,true,true,false><<<blocksPerGrid, threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in.data, dd1_gpu_in.data, dd2_gpu_in.data,
	accumulated_density_gpu, dxyz_gpu_accum.data, dd1_gpu_accum.data, dd2_gpu_accum.data);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 101;
    const int functionalCorrelation = 130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    // Calculate exc_corr and y2a
    libxc_exchange_correlation_cpu<double, true, true, false> (&libxcProxy,
	energy_gpu_in,
	factor_gpu_in,
	number_of_points,
	accumulated_density_gpu,
	dxyz_gpu_accum.data,
        dd1_gpu_accum.data,
	dd2_gpu_accum.data);

    // Join the results.
    gpu_accumulate_energy_and_forces_from_libxc<double, true, true, false><<<blocksPerGrid, threadsPerBlock>>> (
	energy_gpu_in,
	factor_gpu_in,
	point_weights_gpu_in,
	number_of_points,
	accumulated_density_gpu);

    // Free memory
    hipFree(energy_gpu_in);
    hipFree(factor_gpu_in);
    hipFree(point_weights_gpu_in);
    hipFree(partial_density_gpu_in);
    hipFree(accumulated_density_gpu);

}

void accumulate_data_for_libxc_test0005()
{
    printf("** accumulate_data_for_libxc_test0005 **\n");

    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu;

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum;

    dxyz_gpu.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu = NULL;
    double *partial_density_gpu = NULL;
    // Accum
    double *accumulated_density_gpu = NULL;

    // Now the arrays for energy, factors, point_weight and partial_density
    double *energy_gpu = NULL;
    double *factor_gpu = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);

    err = hipMalloc((void**)&energy_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    err = hipMalloc((void**)&point_weights_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu!\n");
    }

    err = hipMalloc((void**)&accumulated_density_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Set the cuda array values to a default value.
    hipMemset(energy_gpu, 0, size);
    hipMemset(factor_gpu, 0, size);
    hipMemset(point_weights_gpu, 1, size);
    hipMemset(partial_density_gpu, 1, size);
    hipMemset(accumulated_density_gpu, 1, size);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    ///////////////////////////////////////////////////
    // Calculate exc_corr and y2a using LIBXC GPU
    libxc_exchange_correlation_gpu<double, true, true, false> (&libxcProxy,
	energy_gpu,
	factor_gpu,
	number_of_points,
	accumulated_density_gpu,
	dxyz_gpu_accum.data,
        dd1_gpu_accum.data,
	dd2_gpu_accum.data);

    ///////////////////////////////////////////////////
    // Check and print the results.
    // Copy back the results before print.
    // Allocate the host input vectors
    double *energy_cpu = (double *)malloc(size);
    double *factor_cpu = (double *)malloc(size);
    double *point_weights_cpu = (double *)malloc(size);
    double *partial_density_cpu = (double *)malloc(size);

    // Copy the vectors from gpu to cpu
    // Be aware that energy_gpu can be NULL.
    err = hipMemcpy(energy_cpu, energy_gpu, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector energy_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(factor_cpu, factor_gpu, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector factor_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(point_weights_cpu, point_weights_gpu, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector point_weights_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(partial_density_cpu, partial_density_gpu, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector partial_density_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    // Before print the results, get the data from cuda
    G2G::vec_type<double,4>* dxyz_cpu;
    G2G::vec_type<double,4>* dd1_cpu;
    G2G::vec_type<double,4>* dd2_cpu;

    // Alloc memory in the host for the gpu data
    uint cpu_vecs_size = number_of_points * sizeof(G2G::vec_type<double,4>);
    dxyz_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);
    dd1_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);
    dd2_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);

    // Copy data from device to host.
    err = hipMemcpy(dxyz_cpu, dxyz_gpu.data, cpu_vecs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dxyz_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dd1_cpu, dd1_gpu.data, cpu_vecs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dd1_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dd2_cpu, dd2_gpu.data, cpu_vecs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dd2_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (dxyz_cpu, dd1_cpu, dd2_cpu, energy_cpu, 
	factor_cpu, NULL, NULL, number_of_points);

    free(energy_cpu);
    free(factor_cpu);
    free(point_weights_cpu);
    free(partial_density_cpu);

    free(dxyz_cpu);
    free(dd1_cpu);
    free(dd2_cpu);

    hipFree(energy_gpu);
    hipFree(factor_gpu);
    hipFree(point_weights_gpu);
    hipFree(partial_density_gpu);


    // Free memory
    hipFree(energy_gpu);
    hipFree(factor_gpu);
    hipFree(point_weights_gpu);
    hipFree(partial_density_gpu);
    hipFree(accumulated_density_gpu);

}

void accumulate_data_for_libxc_test0006()
{
    printf("** accumulate_data_for_libxc_test0006() **\n");

    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in;

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum;
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum;

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *accumulated_density_gpu = NULL;

    // Now the arrays for energy, factors, point_weight and partial_density
    double *energy_gpu_in = NULL;
    double *factor_gpu_in = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);

    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&accumulated_density_gpu, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Set the cuda array values to a default value.
    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemset(point_weights_gpu_in, 1, size);
    hipMemset(partial_density_gpu_in, 1, size);
    hipMemset(accumulated_density_gpu, 1, size);

    // Launch the CUDA Kernel
    int numElements = number_of_points;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    //////////////////////////////////////////////
    // Call the CUDA KERNEL for accumulate point
    gpu_accumulate_point_for_libxc<double,true,true,false><<<blocksPerGrid, threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in.data, dd1_gpu_in.data, dd2_gpu_in.data,
	accumulated_density_gpu, dxyz_gpu_accum.data, dd1_gpu_accum.data, dd2_gpu_accum.data);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    //////////////////////////////////////////////////
    // Calculate exc_corr and y2a in GPU with LIBXC
    libxc_exchange_correlation_gpu<double, true, true, false> (&libxcProxy,
	energy_gpu_in,
	factor_gpu_in,
	number_of_points,
	accumulated_density_gpu,
	dxyz_gpu_accum.data,
        dd1_gpu_accum.data,
	dd2_gpu_accum.data);


    ////////////////////////////////
    // Join the results for LIO
    gpu_accumulate_energy_and_forces_from_libxc<double, true, true, false><<<blocksPerGrid, threadsPerBlock>>> (
	energy_gpu_in,
	factor_gpu_in,
	point_weights_gpu_in,
	number_of_points,
	accumulated_density_gpu);

    ///////////////////////////////////////
    // Check and print the results.
    // Copy back the results before print.
    // Allocate the host input vectors
    double *energy_cpu = (double *)malloc(size);
    double *factor_cpu = (double *)malloc(size);
    double *point_weights_cpu = (double *)malloc(size);
    double *partial_density_cpu = (double *)malloc(size);

    // Copy the vectors from gpu to cpu
    // Be aware that energy_gpu can be NULL.
    err = hipMemcpy(energy_cpu, energy_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector energy_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(factor_cpu, factor_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector factor_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(point_weights_cpu, point_weights_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector point_weights_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(partial_density_cpu, partial_density_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector partial_density_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    // Before print the results, get the data from cuda
    G2G::vec_type<double,4>* dxyz_cpu;
    G2G::vec_type<double,4>* dd1_cpu;
    G2G::vec_type<double,4>* dd2_cpu;

    // Alloc memory in the host for the gpu data
    uint cpu_vecs_size = number_of_points * sizeof(G2G::vec_type<double,4>);
    dxyz_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);
    dd1_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);
    dd2_cpu = (G2G::vec_type<double,4> *)malloc(cpu_vecs_size);

    // Copy data from device to host.
    err = hipMemcpy(dxyz_cpu, dxyz_gpu_in.data, cpu_vecs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dxyz_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dd1_cpu, dd1_gpu_in.data, cpu_vecs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dd1_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dd2_cpu, dd2_gpu_in.data, cpu_vecs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector dd2_gpu from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (dxyz_cpu, dd1_cpu, dd2_cpu, energy_cpu, 
	factor_cpu, point_weights_cpu, 
	partial_density_cpu, number_of_points);

    ////////////////////////////
    // Free Memory
    free(energy_cpu);
    free(factor_cpu);
    free(point_weights_cpu);
    free(partial_density_cpu);

    free(dxyz_cpu);
    free(dd1_cpu);
    free(dd2_cpu);

    hipFree(energy_gpu_in);
    hipFree(factor_gpu_in);
    hipFree(point_weights_gpu_in);
    hipFree(partial_density_gpu_in);
    hipFree(accumulated_density_gpu);

}


///////////////////////////////////////////////////
// Cuda Matrix Helpers
G2G::HostMatrix< G2G::vec_type<double,4> > createMatrix(int size)
{
    //typedef G2G::vec_type<float,4> vec_type4;
    //G2G::CudaMatrix<scalar_type> function_values;
    //G2G::CudaMatrix<vec_type4> gradient_values;
    //G2G::CudaMatrix<vec_type4> hessian_values_transposed;

    //G2G::CudaMatrix< G2G::vec_type<double,4> > aCudaMatrix;
    G2G::HostMatrix< G2G::vec_type<double,4> > aHostMatrix(size, 1);

    //G2G::vec_type<float,4> one(1,1,1,1);
    G2G::vec_type<float,4> zero(0,0,0,0);

    for (int i=0; i<5; i++){
	aHostMatrix(i).x = zero.x;
	aHostMatrix(i).y = zero.y;
	aHostMatrix(i).z = zero.z;
    }

    //aCudaMatrix = aHostMatrix;
    return aHostMatrix;
}


/////////////////////////////////////////////////
//
void accumulate_data_for_libxc_test0007() {
    printf("accumulate_data_for_libxc_test0007()\n");
    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in = createMatrix(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in = createMatrix(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in = createMatrix(number_of_points);

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum = createMatrix(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum = createMatrix(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum = createMatrix(number_of_points);

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *partial_density_gpu_accum = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);
    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_accum, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Now the arrays for energy, factors
    double *energy_gpu_in = NULL;
    double *factor_gpu_in = NULL;
    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    // Launch the CUDA Kernel
    int numElements = n+m;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    ///////////////////////
    // Set data
    //=========
    //= Data  =
    //=========
    //number_of_points:221
    //block_height:1
    //double dxyz[221] = {(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000001,-0.000001,0.000001),(0.000000,-0.000000,0.000001),(0.000000,-0.000001,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000001,0.000001),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000035,-0.000017,0.000026),(0.000009,0.000001,0.000028),(0.000013,0.000007,0.000010),(0.000003,0.000003,0.000001),(0.000002,0.000004,0.000005),(-0.000000,0.000017,-0.000003),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000008,0.000000,0.000008),(0.000002,0.000004,0.000002),(0.000007,-0.000007,0.000007),(0.000002,0.000005,0.000004),(0.000005,-0.000005,0.000009),(0.000005,-0.000009,0.000005),(0.000004,0.000004,0.000002),(0.000009,-0.000005,0.000005),(0.000002,0.000004,0.000007),(0.000003,-0.000003,0.000011),(0.000003,-0.000011,0.000003),(0.000006,0.000004,0.000001),(0.000010,-0.000003,0.000003),(0.000008,-0.000008,0.000004),(0.000005,0.000004,0.000005),(0.000008,-0.000004,0.000008),(0.000004,-0.000008,0.000008),(0.000010,0.000001,0.000003),(0.000003,0.000001,0.000010),(0.000009,-0.000001,0.000006),(0.000008,0.000002,0.000005),(0.000006,-0.000002,0.000010),(0.000005,0.000002,0.000008),(0.000025,0.000007,0.000024),(0.000003,0.000011,0.000002),(0.000026,-0.000025,0.000025),(0.000001,0.000019,0.000006),(0.000019,-0.000018,0.000034),(0.000055,-0.000034,0.000067),(0.000019,-0.000035,0.000019),(0.000007,0.000017,-0.000000),(0.000034,-0.000017,0.000018),(0.000001,0.000018,0.000015),(0.000010,-0.000008,0.000037),(0.000011,-0.000039,0.000010),(0.000013,0.000017,-0.000002),(0.000036,-0.000007,0.000008),(0.000025,-0.000013,0.000025),(0.000030,-0.000028,0.000012),(0.000012,0.000019,0.000010),(0.000029,-0.000010,0.000028),(0.000011,-0.000005,0.000017),(0.000013,-0.000029,0.000030),(0.000029,0.000008,0.000006),(0.000009,0.000007,0.000031),(0.000033,-0.000001,0.000018),(0.000022,0.000014,0.000010),(0.000020,-0.000002,0.000033),(0.000013,0.000014,0.000023),(0.000031,0.000025,0.000028),(-0.000001,0.000014,-0.000001),(0.000040,-0.000035,0.000037),(-0.000007,0.000036,-0.000002),(0.000026,-0.000020,0.000053),(0.000077,-0.000088,0.000098),(0.000028,-0.000053,0.000025),(0.000001,0.000031,-0.000009),(0.000053,-0.000018,0.000021),(0.000007,0.000042,0.000003),(0.000043,-0.000007,0.000040),(0.000036,0.000028,-0.000003),(0.000004,0.000027,0.000039),(0.000047,0.000011,0.000020),(0.000024,0.000037,0.000004),(0.000025,0.000010,0.000046),(0.000010,0.000037,0.000023),(0.000027,0.000054,0.000019),(-0.000008,0.000019,-0.000007),(0.000041,-0.000028,0.000034),(-0.000018,0.000055,-0.000013),(0.000021,-0.000008,0.000054),(0.000066,-0.000120,0.000090),(0.000024,-0.000052,0.000017),(-0.000008,0.000046,-0.000019),(0.000057,-0.000004,0.000011),(-0.000003,0.000069,-0.000010),(0.000044,0.000011,0.000037),(0.000048,0.000036,0.000009),(0.000018,0.000066,-0.000011),(0.000019,0.000034,0.000044),(-0.000001,0.000067,0.000014),(0.000023,0.000084,0.000011),(-0.000016,0.000035,-0.000016),(0.000037,-0.000014,0.000026),(-0.000009,0.000097,-0.000018),(0.000042,0.000035,0.000030),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000009,-0.000018,0.000016),(0.000005,0.000007,0.000001),(0.000011,0.000005,0.000018)};
    //double dd1[221] = {(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,-0.000000),(-0.000000,0.000000,0.000000),(-0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000001,0.000001,0.000001),(0.000000,-0.000000,0.000001),(0.000000,0.000002,0.000000),(0.000001,-0.000000,0.000000),(-0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(-0.000000,0.000001,0.000001),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000014,-0.000019,-0.000005),(-0.000011,-0.000016,0.000027),(0.000007,-0.000005,0.000001),(0.000003,0.000002,-0.000002),(-0.000002,0.000002,0.000003),(-0.000004,0.000095,0.000018),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,-0.000000,0.000000),(-0.000000,-0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,-0.000000,-0.000000),(0.000000,-0.000000,-0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,-0.000000),(-0.000000,0.000000,0.000000),(-0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,-0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000001),(0.000000,0.000001,0.000000),(0.000001,0.000000,0.000000),(0.000000,0.000000,0.000001),(0.000000,0.000000,0.000001),(0.000000,0.000001,0.000000),(0.000001,0.000000,-0.000000),(0.000001,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,-0.000000,0.000001),(-0.000000,-0.000000,0.000001),(-0.000000,0.000001,-0.000000),(0.000001,-0.000000,-0.000000),(0.000001,-0.000000,-0.000000),(0.000001,-0.000000,0.000000),(0.000000,-0.000000,0.000001),(0.000001,-0.000000,0.000000),(0.000001,-0.000000,0.000000),(0.000001,0.000000,-0.000000),(-0.000000,0.000001,0.000000),(-0.000000,0.000000,0.000001),(0.000000,-0.000000,0.000001),(0.000000,-0.000000,0.000001),(0.000000,0.000001,-0.000000),(0.000007,-0.000004,0.000007),(-0.000000,0.000006,-0.000000),(0.000005,0.000004,0.000004),(-0.000002,0.000003,0.000002),(0.000001,0.000001,0.000011),(0.000001,0.000011,0.000001),(0.000002,0.000003,-0.000002),(0.000011,0.000001,0.000001),(-0.000002,-0.000000,0.000008),(-0.000002,-0.000002,0.000016),(-0.000002,0.000016,-0.000002),(0.000007,0.000000,-0.000003),(0.000016,-0.000002,-0.000002),(0.000007,0.000007,-0.000001),(0.000003,0.000000,0.000002),(0.000007,-0.000002,0.000007),(-0.000001,0.000007,0.000007),(0.000014,-0.000004,-0.000002),(-0.000001,-0.000004,0.000015),(0.000013,-0.000003,0.000002),(0.000009,-0.000002,0.000001),(0.000003,-0.000003,0.000013),(0.000001,-0.000003,0.000009),(0.000006,-0.000021,0.000003),(-0.000001,0.000033,0.000004),(0.000003,-0.000001,0.000001),(-0.000012,0.000018,-0.000012),(-0.000009,-0.000012,0.000021),(0.000108,0.000012,0.000166),(-0.000008,0.000020,-0.000010),(-0.000008,0.000020,-0.000010),(0.000022,-0.000013,-0.000011),(-0.000016,0.000001,-0.000004),(-0.000017,-0.000021,0.000032),(-0.000017,0.000034,-0.000019),(-0.000002,0.000006,-0.000014),(0.000031,-0.000021,-0.000019),(0.000058,0.000034,0.000070),(0.000011,0.000006,-0.000017),(-0.000008,0.000002,-0.000012),(0.000010,-0.000019,0.000007),(0.000028,0.000035,0.000056),(-0.000016,0.000008,0.000010),(0.000021,-0.000019,-0.000019),(-0.000016,-0.000021,0.000022),(0.000023,-0.000023,-0.000008),(0.000007,-0.000011,-0.000014),(-0.000005,-0.000023,0.000021),(-0.000010,-0.000013,0.000005),(-0.000018,-0.000049,-0.000029),(0.000003,0.000100,0.000020),(-0.000023,-0.000037,-0.000030),(-0.000023,0.000058,-0.000029),(-0.000041,-0.000054,0.000001),(0.000149,0.000116,0.000241),(-0.000040,-0.000004,-0.000046),(-0.000022,0.000066,-0.000011),(0.000008,-0.000054,-0.000048),(-0.000032,0.000016,-0.000039),(-0.000012,-0.000062,-0.000021),(0.000003,-0.000041,-0.000050),(-0.000046,-0.000047,-0.000002),(0.000007,-0.000061,-0.000044),(-0.000016,-0.000018,-0.000045),(-0.000035,-0.000062,-0.000001),(-0.000038,-0.000023,-0.000025),(-0.000040,-0.000078,-0.000057),(0.000006,0.000207,0.000035),(-0.000058,-0.000088,-0.000072),(-0.000028,0.000128,-0.000030),(-0.000083,-0.000109,-0.000022),(0.000146,0.000282,0.000245),(-0.000084,-0.000038,-0.000096),(-0.000027,0.000142,-0.000004),(-0.000007,-0.000109,-0.000095),(-0.000047,0.000050,-0.000053),(-0.000038,-0.000118,-0.000055),(-0.000003,-0.000105,-0.000083),(-0.000026,-0.000015,-0.000070),(-0.000070,-0.000109,-0.000020),(-0.000064,-0.000024,-0.000042),(-0.000057,-0.000111,-0.000077),(-0.000002,0.000337,0.000033),(-0.000100,-0.000144,-0.000121),(-0.000057,0.000094,-0.000062),(-0.000065,-0.000183,-0.000088),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,0.000000),(-0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(-0.000004,0.000013,0.000007),(0.000001,0.000011,-0.000001),(-0.000000,-0.000009,0.000014)};
    //double dd2[221] = {(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000001,0.000001,-0.000001),(-0.000000,0.000001,-0.000000),(-0.000001,0.000000,-0.000001),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000001),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000017,0.000027,-0.000011),(0.000003,0.000011,0.000005),(0.000010,0.000011,0.000009),(0.000005,0.000000,0.000001),(0.000003,0.000002,0.000007),(-0.000028,0.000011,-0.000050),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000001,0.000010,0.000001),(0.000002,0.000001,0.000002),(-0.000008,0.000008,-0.000008),(0.000003,0.000002,0.000006),(-0.000005,0.000008,-0.000008),(-0.000008,0.000005,-0.000008),(0.000006,0.000001,0.000003),(-0.000008,0.000008,-0.000004),(0.000002,0.000002,0.000007),(-0.000002,0.000006,-0.000006),(-0.000006,0.000002,-0.000006),(0.000007,0.000001,0.000002),(-0.000005,0.000005,-0.000001),(-0.000011,0.000005,-0.000005),(0.000005,0.000005,0.000005),(-0.000005,0.000011,-0.000005),(-0.000005,0.000005,-0.000011),(0.000002,0.000005,0.000001),(0.000001,0.000006,0.000002),(-0.000002,0.000010,-0.000001),(0.000005,0.000007,0.000003),(-0.000001,0.000010,-0.000002),(0.000003,0.000008,0.000004),(0.000011,0.000022,0.000012),(-0.000008,0.000006,-0.000012),(-0.000021,0.000022,-0.000020),(0.000007,-0.000005,0.000015),(-0.000010,0.000022,-0.000020),(-0.000080,0.000147,-0.000094),(-0.000022,0.000011,-0.000021),(0.000013,-0.000005,0.000001),(-0.000019,0.000020,-0.000008),(0.000007,-0.000006,0.000026),(-0.000000,0.000012,-0.000008),(-0.000013,0.000002,-0.000012),(0.000022,-0.000008,0.000003),(-0.000006,0.000008,0.000002),(-0.000056,0.000071,-0.000057),(-0.000029,0.000011,-0.000009),(0.000020,0.000001,0.000020),(-0.000009,0.000028,-0.000008),(-0.000036,0.000043,-0.000048),(-0.000012,0.000012,-0.000029),(0.000016,0.000004,0.000008),(0.000007,0.000008,0.000015),(0.000002,0.000021,0.000003),(0.000022,0.000007,0.000014),(0.000002,0.000023,0.000002),(0.000014,0.000010,0.000022),(0.000034,0.000015,0.000036),(-0.000039,0.000016,-0.000054),(-0.000021,0.000024,-0.000017),(-0.000001,-0.000014,0.000013),(-0.000003,0.000022,-0.000013),(-0.000189,0.000199,-0.000226),(-0.000022,0.000007,-0.000018),(0.000009,-0.000009,-0.000019),(-0.000014,0.000018,0.000003),(0.000035,-0.000013,0.000032),(0.000002,0.000031,0.000005),(0.000045,-0.000013,0.000021),(0.000021,-0.000005,0.000046),(0.000022,0.000016,0.000021),(0.000049,-0.000008,0.000029),(0.000017,0.000021,0.000023),(0.000032,-0.000003,0.000050),(0.000060,0.000018,0.000061),(-0.000067,0.000023,-0.000096),(-0.000019,0.000026,-0.000011),(-0.000014,-0.000012,0.000008),(0.000008,0.000023,-0.000004),(-0.000262,0.000177,-0.000321),(-0.000021,0.000001,-0.000012),(0.000009,-0.000006,-0.000047),(-0.000007,0.000016,0.000017),(0.000050,-0.000013,0.000041),(0.000016,0.000038,0.000021),(0.000046,0.000016,0.000040),(0.000077,-0.000010,0.000040),(0.000036,0.000023,0.000049),(0.000048,-0.000006,0.000079),(0.000085,0.000042,0.000083),(-0.000070,0.000022,-0.000113),(-0.000028,0.000039,-0.000017),(0.000072,0.000003,0.000055),(0.000023,0.000061,0.000029),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000011,0.000010,-0.000019),(0.000004,0.000001,-0.000003),(0.000006,0.000014,0.000009)};
    //double energy[221] = {1.542830e-42,0.000000e+00,1.017876e-16,4.577482e-41,2.802597e-45,0.000000e+00,2.620864e-16,4.577482e-41,2.620872e-16,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,2.003857e-43,0.000000e+00,-2.317951e+13,1.926055e-04,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,7.229655e-17,4.577482e-41,7.230719e-17,4.577482e-41,0.000000e+00,0.000000e+00,7.230781e-17,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,1.401298e-45,1.401298e-45,1.401298e-45,1.401298e-45,4.577482e-41,6.532545e-27,4.577482e-41,-3.194302e-38,5.898247e-02,4.237519e-37,9.809089e-45,3.363116e-44,0.000000e+00,2.025024e-08,0.000000e+00,2.848895e-08,1.511689e-09,2.150964e-08,0.000000e+00,1.541544e-08,1.854911e-07,2.257209e-08,0.000000e+00,1.605298e-08,7.123480e-09,2.104863e-07,0.000000e+00,9.799036e-05,1.466930e-04,2.309945e-04,0.000000e+00,1.562906e-04,1.148952e-04,1.203459e-04,0.000000e+00,3.012822e-04,4.234389e-05,2.005827e-04,0.000000e+00,8.251942e-05,1.142085e-04,8.502679e-05,0.000000e+00,3.224969e-04,1.431179e-04,3.734327e-04,0.000000e+00,3.013988e-03,1.226029e-03,2.129179e-02,0.000000e+00,1.464419e-02,5.501955e-03,1.881593e-02,0.000000e+00,1.660532e-02,7.767329e-03,9.980457e-03,0.000000e+00,1.278249e-02,1.585113e-03,1.794737e-02,0.000000e+00,1.581253e-02,5.122344e-03,4.728785e-03,-7.521544e-06,-3.115620e+05,7.524181e+01,2.789771e-02,-2.056223e-05,1.973437e+27,4.870370e+01,3.909907e-02,-2.700995e-05,8.916755e-01,6.818591e+01,3.266954e-02,-3.893143e-06,2.683526e-27,2.092911e-23,8.694591e-26,0.000000e+00,9.707691e-21,4.164478e-27,5.105821e-26,0.000000e+00,5.618195e-22,8.011395e-23,7.939312e-27,0.000000e+00,1.583467e-42,1.401298e-45,1.014504e-16,4.577482e-41,2.802597e-45,0.000000e+00,2.620864e-16,4.577482e-41,2.620872e-16,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,2.003857e-43,0.000000e+00,-3.194302e-38,5.898247e-02,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,6.283977e-17,4.577482e-41,6.284513e-17,4.577482e-41,0.000000e+00,0.000000e+00,6.284652e-17,4.577482e-41,0.000000e+00,0.000000e+00,1.399897e-42,1.401298e-45,1.401298e-45,1.401298e-45,1.401298e-45,4.577482e-41,4.237290e-37,9.809089e-45,-2.317951e+13,1.926055e-04,4.237290e-37,9.809089e-45,3.363116e-44,0.000000e+00,2.088217e-27,0.000000e+00,1.568769e-26,6.419391e-27,3.931566e-23,0.000000e+00,6.554724e-27,3.067580e-27,2.328027e-26,0.000000e+00,1.053539e-26,5.014781e-27,7.749388e-24,0.000000e+00,8.620366e-28,2.289166e-27,5.293036e-27,0.000000e+00,6.232589e-27,5.673471e-27,2.332169e-26,0.000000e+00,1.177772e-08,2.611032e-09,1.260247e-08,0.000000e+00,3.821657e-08,2.177334e-09,7.358555e-09,0.000000e+00,4.254145e-09,1.708416e-09,7.111237e-09,0.000000e+00,5.839961e-09,8.409097e-09,4.951757e-09,0.000000e+00,2.549411e-09,1.483298e-09,1.624575e-09,0.000000e+00,1.740956e-09};
    //double factor[221] = {1.542830e-42,0.000000e+00,1.017876e-16,4.577482e-41,2.802597e-45,0.000000e+00,2.620864e-16,4.577482e-41,2.620872e-16,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,2.003857e-43,0.000000e+00,-2.317951e+13,1.926055e-04,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,7.229655e-17,4.577482e-41,7.230719e-17,4.577482e-41,0.000000e+00,0.000000e+00,7.230781e-17,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,1.401298e-45,1.401298e-45,1.401298e-45,1.401298e-45,4.577482e-41,6.530166e-27,4.577482e-41,-3.194302e-38,5.898247e-02,4.237519e-37,9.809089e-45,3.363116e-44,0.000000e+00,6.836695e-12,1.675647e-11,6.620036e-12,6.265301e-12,6.394112e-12,6.476615e-12,7.319163e-12,3.605303e-11,6.644645e-12,6.467787e-12,6.112196e-12,5.683108e-12,6.599283e-12,6.844061e-12,6.563489e-12,6.017392e-12,6.399866e-12,6.154383e-12,3.026520e-07,3.557238e-07,3.342354e-07,3.519032e-07,3.405898e-07,3.512081e-07,3.150973e-07,3.629471e-07,3.268906e-07,3.745598e-07,3.230706e-07,3.582586e-07,3.422037e-07,3.462203e-07,3.601238e-07,3.121515e-07,3.722795e-07,3.343488e-07,3.569448e-07,3.585053e-07,3.710899e-07,3.638215e-07,3.261631e-07,3.464344e-07,3.334570e-07,4.756184e-05,4.773591e-05,4.515327e-05,4.688610e-05,4.413019e-05,4.858421e-05,4.693521e-05,5.063206e-05,4.833565e-05,4.781871e-05,4.642509e-05,4.634951e-05,5.089617e-05,4.829466e-05,5.122057e-05,3.713917e-05,4.961211e-05,4.833119e-05,3.915227e-05,4.985524e-05,4.954117e-05,4.686260e-04,4.303385e-04,3.447189e-04,4.114783e-04,4.361123e-04,4.445273e-04,4.612404e-04,4.645406e-04,4.760396e-04,4.713000e-04,4.726021e-04,4.226256e-04,4.610711e-04,4.390908e-04,1.583467e-42,1.401298e-45,1.014504e-16,4.577482e-41,2.802597e-45,0.000000e+00,2.620864e-16,4.577482e-41,2.620872e-16,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,2.003857e-43,0.000000e+00,-3.194302e-38,5.898247e-02,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,6.283977e-17,4.577482e-41,6.284513e-17,4.577482e-41,0.000000e+00,0.000000e+00,6.284652e-17,4.577482e-41,0.000000e+00,0.000000e+00,1.399897e-42,1.401298e-45,1.401298e-45,1.401298e-45,1.401298e-45,4.577482e-41,4.237290e-37,9.809089e-45,-2.317951e+13,1.926055e-04,4.237290e-37,9.809089e-45,3.363116e-44,0.000000e+00,1.856939e-05,4.965303e-05,7.526137e-05,6.964390e-07,1.641322e-04,3.707860e-06,2.637519e-04,2.127130e-04,2.210860e-05,1.228141e-07,5.726594e-04,4.518460e-04,1.110597e-05,1.670070e-05,5.406669e-04,6.669017e-04,8.500725e-05,6.133204e-04,2.710157e-04,1.765020e-04,3.489698e-05,4.957735e-04,4.268136e-04,2.140554e-22,2.745518e-24,1.099713e-26,1.039016e-25,7.914708e-22,1.839824e-20,9.827770e-23,8.790141e-24,4.658280e-18,2.503744e-16,3.010181e-19,5.753074e-22,6.707894e-21,5.924780e-21,1.547127e-26,1.484105e-24,6.045916e-25,5.328845e-06,2.523798e-05,1.002050e-05};
    //double point_weights[221] = {0.000000e+00,0.000000e+00,6.356219e-06,3.324887e-04,3.143648e-02,3.212402e-01,1.299464e-05,7.277725e-04,0.000000e+00,2.066700e+00,2.379370e-01,5.970362e-02,6.884030e-05,1.351831e-03,7.788771e-02,2.020641e-03,1.322531e-06,3.924060e-06,1.305768e-06,1.925648e-04,1.586370e-02,3.594067e-04,1.449458e-01,7.987326e-06,4.113678e-04,0.000000e+00,1.094970e-01,3.780463e-02,0.000000e+00,5.679121e-06,0.000000e+00,2.230565e-04,1.453141e-02,4.649304e-04,0.000000e+00,8.580421e-02,3.249447e-02,0.000000e+00,1.818959e-05,1.134347e-03,1.042623e-01,4.639464e-02,6.318915e-03,1.360220e+02,1.326793e+02,1.313792e+02,1.313792e+02,1.313792e+02,1.227952e+02,1.227954e+02,1.227835e+02,1.335201e+02,1.335201e+02,1.335200e+02,9.516099e+01,9.764464e+01,9.107944e+01,9.713596e+01,1.196765e+02,1.200841e+02,1.316433e+02,1.316505e+02,1.316438e+02,1.316480e+02,1.316498e+02,1.316524e+02,1.772504e+01,1.728354e+01,1.711418e+01,1.711418e+01,1.711418e+01,1.599609e+01,1.599612e+01,1.599554e+01,1.739307e+01,1.739307e+01,1.739307e+01,1.258319e+01,1.272966e+01,1.273444e+01,1.230952e+01,1.270183e+01,1.563527e+01,1.565721e+01,1.714936e+01,1.706164e+01,1.714969e+01,1.714966e+01,1.714972e+01,1.714966e+01,1.709727e+01,1.714977e+01,3.601541e+00,3.511343e+00,3.476937e+00,3.476937e+00,3.476937e+00,3.212072e+00,3.249791e+00,3.249793e+00,3.174251e+00,3.249757e+00,3.533596e+00,3.504852e+00,3.533596e+00,3.533596e+00,2.576064e+00,2.586982e+00,2.587315e+00,2.553255e+00,2.585052e+00,3.180675e+00,3.182183e+00,3.484143e+00,3.477949e+00,3.484163e+00,3.484166e+00,3.484167e+00,3.484161e+00,3.480554e+00,3.484168e+00,9.663483e-01,8.954918e-01,9.421041e-01,9.202220e-01,9.328728e-01,9.328728e-01,9.153517e-01,9.328728e-01,8.686274e-01,8.719293e-01,8.719295e-01,8.649900e-01,8.719268e-01,9.480746e-01,9.456108e-01,9.480746e-01,9.480746e-01,8.537841e-01,8.539019e-01,9.348111e-01,9.343160e-01,9.348124e-01,9.345296e-01,3.132776e-01,3.003089e-01,3.054141e-01,3.011668e-01,3.024214e-01,2.675158e-01,3.024214e-01,3.006469e-01,3.024214e-01,2.823578e-01,2.826646e-01,2.826646e-01,2.819968e-01,2.826644e-01,2.886273e-01,3.073496e-01,3.071240e-01,3.073496e-01,2.925198e-01,3.073496e-01,2.768208e-01,2.768309e-01,3.030502e-01,3.030071e-01,3.030503e-01,3.030261e-01,1.163402e-01,1.128603e-01,1.134196e-01,1.121797e-01,1.123082e-01,1.077705e-01,1.123082e-01,1.121239e-01,1.123082e-01,1.141165e-01,1.141384e-01,1.028049e-01,1.028059e-01,1.125418e-01,1.125377e-01,1.125418e-01,1.125395e-01,4.780666e-02,4.654431e-02,4.660649e-02,4.613614e-02,4.614981e-02,4.558606e-02,4.614981e-02,4.613001e-02,4.614981e-02,4.689961e-02,4.690185e-02,4.624578e-02,4.624538e-02,4.624578e-02,4.624556e-02,2.122287e-02,2.068298e-02,2.069007e-02,2.082095e-02,2.082120e-02,2.571491e-07,3.053828e-05,3.673155e-03,1.025607e-02,0.000000e+00,0.000000e+00,1.847405e-06,1.768821e-07,0.000000e+00,0.000000e+00,4.100356e-04,8.746948e-10,1.113151e-13,4.551340e-11};
    //double partial_densities[221] = {1.016692e-33,2.333626e-34,8.367814e-34,6.744978e-35,4.493371e-36,4.396106e-37,1.908333e-34,4.848228e-35,7.228556e-34,1.717567e-38,6.815741e-37,2.776831e-36,1.143339e-34,3.467837e-35,2.173183e-36,2.919116e-35,3.249745e-13,1.280894e-13,2.872959e-13,5.841356e-14,1.052731e-14,4.778827e-14,2.419135e-15,1.127769e-13,4.739891e-14,2.618893e-13,3.192557e-15,6.648584e-15,4.077459e-07,2.149901e-07,3.746791e-07,1.251948e-07,3.841547e-08,1.084211e-07,3.515827e-07,1.679403e-08,2.794462e-08,3.479711e-05,2.007635e-05,1.120590e-05,2.301993e-06,3.572237e-06,1.566220e-05,5.111461e-31,5.111677e-31,5.111461e-31,5.111597e-31,5.111461e-31,5.111840e-31,5.111245e-31,5.111542e-31,5.111677e-31,5.111786e-31,5.111677e-31,5.111892e-31,5.111867e-31,5.111594e-31,5.111542e-31,5.111461e-31,5.111461e-31,5.111489e-31,5.111542e-31,5.111299e-31,5.111867e-31,5.111786e-31,5.111299e-31,1.202872e-12,1.203062e-12,1.203132e-12,1.203147e-12,1.203080e-12,1.203036e-12,1.203051e-12,1.202950e-12,1.203056e-12,1.203025e-12,1.203067e-12,1.201019e-12,1.202777e-12,1.202918e-12,1.198146e-12,1.202207e-12,1.202323e-12,1.202721e-12,1.202982e-12,1.201824e-12,1.203016e-12,1.203111e-12,1.203109e-12,1.203064e-12,1.202214e-12,1.203094e-12,1.056859e-07,1.069813e-07,1.068918e-07,1.069731e-07,1.067879e-07,9.925203e-08,1.065169e-07,1.067720e-07,9.567351e-08,1.061269e-07,1.068738e-07,1.002584e-07,1.067610e-07,1.069669e-07,1.020017e-07,1.054180e-07,1.061756e-07,9.844285e-08,1.040931e-07,1.043744e-07,1.052831e-07,1.062094e-07,1.033102e-07,1.064327e-07,1.068430e-07,1.067436e-07,1.064088e-07,1.040096e-07,1.067188e-07,5.449067e-06,2.706551e-06,5.836964e-06,3.695634e-06,5.801161e-06,5.833626e-06,3.451926e-06,5.762271e-06,4.397482e-06,5.673675e-06,5.758306e-06,3.997644e-06,5.560026e-06,5.793905e-06,4.525559e-06,5.752838e-06,5.831018e-06,5.168859e-06,5.356980e-06,5.582546e-06,4.978430e-06,5.640204e-06,5.102124e-06,2.601504e-05,9.496978e-06,3.058765e-05,1.358994e-05,3.008526e-05,2.710820e-05,3.054102e-05,1.249342e-05,2.956473e-05,1.764504e-05,2.846226e-05,2.951743e-05,1.532756e-05,2.717309e-05,1.355043e-05,2.998640e-05,1.852374e-05,2.944374e-05,1.020576e-05,3.050481e-05,2.341930e-05,2.510869e-05,2.741899e-05,2.183935e-05,2.806830e-05,2.283266e-05,5.046140e-05,1.499352e-05,6.507664e-05,2.169488e-05,6.326965e-05,4.041267e-05,6.489926e-05,1.996210e-05,6.149049e-05,3.179689e-05,6.107018e-05,4.349307e-05,4.785749e-05,5.461262e-05,3.951244e-05,5.657595e-05,4.188267e-05,6.643681e-05,1.701268e-05,9.193084e-05,2.527302e-05,8.849833e-05,3.780743e-05,9.155914e-05,2.338526e-05,8.530516e-05,3.873166e-05,8.451808e-05,7.331888e-05,4.973260e-05,7.654892e-05,5.304757e-05,7.487628e-05,1.748799e-05,1.078457e-04,4.236045e-05,9.790303e-05,4.905088e-17,3.745068e-17,4.379298e-19,1.966976e-19,5.755041e-15,2.728153e-14,1.389058e-16,3.278741e-16,4.960657e-15,6.570254e-15,9.327300e-19,1.487219e-05,5.521405e-06,1.366262e-05};

    double partial_densities_cpu[10] = {1.016692e-33,2.333626e-34,8.367814e-34,6.744978e-35,4.493371e-36,4.396106e-37,1.908333e-34,4.848228e-35,7.228556e-34,1.717567e-38};
    double point_weights_cpu[10] = {0.000000e+00,0.000000e+00,6.356219e-06,3.324887e-04,3.143648e-02,3.212402e-01,1.299464e-05,7.277725e-04,0.000000e+00,2.066700e+00};
    double input[number_of_points];
    for (int i=0; i<number_of_points; i++) {
	input[i]=0.001*i;
    }
    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    //hipMemcpy(energy_gpu_in, input, size, hipMemcpyHostToDevice);
    //hipMemcpy(factor_gpu_in, input, size, hipMemcpyHostToDevice);
    hipMemcpy(point_weights_gpu_in, point_weights_cpu, size, hipMemcpyHostToDevice);
    hipMemcpy(partial_density_gpu_in, partial_densities_cpu, size, hipMemcpyHostToDevice);
    //hipMemcpy(dxyz_gpu_in.data, input, size, hipMemcpyHostToDevice);
    //hipMemcpy(dd1_gpu_in.data, input, size, hipMemcpyHostToDevice);
    //hipMemcpy(dd2_gpu_in.data, input, size, hipMemcpyHostToDevice);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    /////////////////////////////////
    // LIBXC VERSION
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    gpu_accumulate_point_for_libxc<double,true,true,false><<<blocksPerGrid,threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in.data, dd1_gpu_in.data, dd2_gpu_in.data,
	partial_density_gpu_accum, dxyz_gpu_accum.data, dd1_gpu_accum.data, dd2_gpu_accum.data);

    // Calculate exc_corr and y2a
    libxc_exchange_correlation_gpu<double, true, true, false> (&libxcProxy,
	energy_gpu_in,
	factor_gpu_in,
	number_of_points,
	partial_density_gpu_accum,
	dxyz_gpu_accum.data,
        dd1_gpu_accum.data,
	dd2_gpu_accum.data);
    
    // Join the results.
    gpu_accumulate_energy_and_forces_from_libxc<double, true, true, false><<<blocksPerGrid, threadsPerBlock>>> (
	energy_gpu_in,
	factor_gpu_in,
	point_weights_gpu_in,
	number_of_points,
	partial_density_gpu_accum);

    ///////////////////////////
    // Print libxc results
    // Allocate the host input vectors
    double *energy_cpu = (double *)malloc(size);
    double *factor_cpu = (double *)malloc(size);

    // Copy the vectors from gpu to cpu
    // Be aware that energy_gpu can be NULL.
    err = hipMemcpy(energy_cpu, energy_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector energy_gpu_in from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(factor_cpu, factor_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector factor_gpu_in from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (NULL, NULL, NULL, energy_cpu,
	factor_cpu, NULL, NULL, number_of_points);

    ////////////////////////////////////////
    // LIO VERSION
    // Now the arrays for energy, factors
    double *energy_gpu_in2 = NULL;
    double *factor_gpu_in2 = NULL;
    err = hipMalloc((void**)&energy_gpu_in2, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu_in2!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in2, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu_in2!\n");
    }

    gpu_accumulate_point<double, true, true, false><<<blocksPerGrid,threadsPerBlock>>> (
	energy_gpu_in2, 
	factor_gpu_in2,
	point_weights_gpu_in,
	number_of_points,
	block_height,
	partial_density_gpu_in,
	dxyz_gpu_in.data,
	dd1_gpu_in.data,
	dd2_gpu_in.data);

    ///////////////////////////////////////
    // Print LIO results.
    // Copy back the results before print.
    // Allocate the host input vectors
    double *energy_cpu2 = (double *)malloc(size);
    double *factor_cpu2 = (double *)malloc(size);

    // Copy the vectors from gpu to cpu
    // Be aware that energy_gpu can be NULL.
    err = hipMemcpy(energy_cpu2, energy_gpu_in2, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector energy_gpu_in2 from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(factor_cpu2, factor_gpu_in2, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector factor_gpu_in2 from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (NULL, NULL, NULL, energy_cpu2,
	factor_cpu2, NULL, NULL, number_of_points);

    ////////////////////////////
    // Free Memory CPU
    free(energy_cpu);
    free(factor_cpu);
    free(energy_cpu2);
    free(factor_cpu2);

    ///////////////////////////
    // Free memory GPU
    hipFree (point_weights_gpu_in);
    hipFree (partial_density_gpu_in);
    hipFree (partial_density_gpu_accum);
    hipFree (energy_gpu_in);
    hipFree (factor_gpu_in);
    hipFree (energy_gpu_in2);
    hipFree (factor_gpu_in2);

}


/////////////////////////////////////////////////
//
void accumulate_data_for_libxc_test0008(int iterations) {
    printf("accumulate_data_for_libxc_test0008()\n");
    hipError_t err = hipSuccess;
    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in = createMatrix(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in = createMatrix(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in = createMatrix(number_of_points);

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum = createMatrix(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum = createMatrix(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum = createMatrix(number_of_points);

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),m);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),m);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *partial_density_gpu_accum = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);
    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_accum, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Now the arrays for energy, factors
    double *energy_gpu_in = NULL;
    double *factor_gpu_in = NULL;
    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    // Launch the CUDA Kernel
    int numElements = n+m;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    ///////////////////////
    // Set data
    //=========
    //= Data  =
    //=========
    //number_of_points:221
    //block_height:1
    //double dxyz[221] = {(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000001,-0.000001,0.000001),(0.000000,-0.000000,0.000001),(0.000000,-0.000001,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000001,0.000001),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000035,-0.000017,0.000026),(0.000009,0.000001,0.000028),(0.000013,0.000007,0.000010),(0.000003,0.000003,0.000001),(0.000002,0.000004,0.000005),(-0.000000,0.000017,-0.000003),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000008,0.000000,0.000008),(0.000002,0.000004,0.000002),(0.000007,-0.000007,0.000007),(0.000002,0.000005,0.000004),(0.000005,-0.000005,0.000009),(0.000005,-0.000009,0.000005),(0.000004,0.000004,0.000002),(0.000009,-0.000005,0.000005),(0.000002,0.000004,0.000007),(0.000003,-0.000003,0.000011),(0.000003,-0.000011,0.000003),(0.000006,0.000004,0.000001),(0.000010,-0.000003,0.000003),(0.000008,-0.000008,0.000004),(0.000005,0.000004,0.000005),(0.000008,-0.000004,0.000008),(0.000004,-0.000008,0.000008),(0.000010,0.000001,0.000003),(0.000003,0.000001,0.000010),(0.000009,-0.000001,0.000006),(0.000008,0.000002,0.000005),(0.000006,-0.000002,0.000010),(0.000005,0.000002,0.000008),(0.000025,0.000007,0.000024),(0.000003,0.000011,0.000002),(0.000026,-0.000025,0.000025),(0.000001,0.000019,0.000006),(0.000019,-0.000018,0.000034),(0.000055,-0.000034,0.000067),(0.000019,-0.000035,0.000019),(0.000007,0.000017,-0.000000),(0.000034,-0.000017,0.000018),(0.000001,0.000018,0.000015),(0.000010,-0.000008,0.000037),(0.000011,-0.000039,0.000010),(0.000013,0.000017,-0.000002),(0.000036,-0.000007,0.000008),(0.000025,-0.000013,0.000025),(0.000030,-0.000028,0.000012),(0.000012,0.000019,0.000010),(0.000029,-0.000010,0.000028),(0.000011,-0.000005,0.000017),(0.000013,-0.000029,0.000030),(0.000029,0.000008,0.000006),(0.000009,0.000007,0.000031),(0.000033,-0.000001,0.000018),(0.000022,0.000014,0.000010),(0.000020,-0.000002,0.000033),(0.000013,0.000014,0.000023),(0.000031,0.000025,0.000028),(-0.000001,0.000014,-0.000001),(0.000040,-0.000035,0.000037),(-0.000007,0.000036,-0.000002),(0.000026,-0.000020,0.000053),(0.000077,-0.000088,0.000098),(0.000028,-0.000053,0.000025),(0.000001,0.000031,-0.000009),(0.000053,-0.000018,0.000021),(0.000007,0.000042,0.000003),(0.000043,-0.000007,0.000040),(0.000036,0.000028,-0.000003),(0.000004,0.000027,0.000039),(0.000047,0.000011,0.000020),(0.000024,0.000037,0.000004),(0.000025,0.000010,0.000046),(0.000010,0.000037,0.000023),(0.000027,0.000054,0.000019),(-0.000008,0.000019,-0.000007),(0.000041,-0.000028,0.000034),(-0.000018,0.000055,-0.000013),(0.000021,-0.000008,0.000054),(0.000066,-0.000120,0.000090),(0.000024,-0.000052,0.000017),(-0.000008,0.000046,-0.000019),(0.000057,-0.000004,0.000011),(-0.000003,0.000069,-0.000010),(0.000044,0.000011,0.000037),(0.000048,0.000036,0.000009),(0.000018,0.000066,-0.000011),(0.000019,0.000034,0.000044),(-0.000001,0.000067,0.000014),(0.000023,0.000084,0.000011),(-0.000016,0.000035,-0.000016),(0.000037,-0.000014,0.000026),(-0.000009,0.000097,-0.000018),(0.000042,0.000035,0.000030),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000009,-0.000018,0.000016),(0.000005,0.000007,0.000001),(0.000011,0.000005,0.000018)};
    //double dd1[221] = {(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,-0.000000),(-0.000000,0.000000,0.000000),(-0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000001,0.000001,0.000001),(0.000000,-0.000000,0.000001),(0.000000,0.000002,0.000000),(0.000001,-0.000000,0.000000),(-0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(-0.000000,0.000001,0.000001),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000014,-0.000019,-0.000005),(-0.000011,-0.000016,0.000027),(0.000007,-0.000005,0.000001),(0.000003,0.000002,-0.000002),(-0.000002,0.000002,0.000003),(-0.000004,0.000095,0.000018),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,-0.000000,0.000000),(-0.000000,-0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,-0.000000,-0.000000),(0.000000,-0.000000,-0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,-0.000000),(-0.000000,0.000000,0.000000),(-0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,-0.000000),(0.000000,-0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000001),(0.000000,0.000001,0.000000),(0.000001,0.000000,0.000000),(0.000000,0.000000,0.000001),(0.000000,0.000000,0.000001),(0.000000,0.000001,0.000000),(0.000001,0.000000,-0.000000),(0.000001,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,-0.000000,0.000001),(-0.000000,-0.000000,0.000001),(-0.000000,0.000001,-0.000000),(0.000001,-0.000000,-0.000000),(0.000001,-0.000000,-0.000000),(0.000001,-0.000000,0.000000),(0.000000,-0.000000,0.000001),(0.000001,-0.000000,0.000000),(0.000001,-0.000000,0.000000),(0.000001,0.000000,-0.000000),(-0.000000,0.000001,0.000000),(-0.000000,0.000000,0.000001),(0.000000,-0.000000,0.000001),(0.000000,-0.000000,0.000001),(0.000000,0.000001,-0.000000),(0.000007,-0.000004,0.000007),(-0.000000,0.000006,-0.000000),(0.000005,0.000004,0.000004),(-0.000002,0.000003,0.000002),(0.000001,0.000001,0.000011),(0.000001,0.000011,0.000001),(0.000002,0.000003,-0.000002),(0.000011,0.000001,0.000001),(-0.000002,-0.000000,0.000008),(-0.000002,-0.000002,0.000016),(-0.000002,0.000016,-0.000002),(0.000007,0.000000,-0.000003),(0.000016,-0.000002,-0.000002),(0.000007,0.000007,-0.000001),(0.000003,0.000000,0.000002),(0.000007,-0.000002,0.000007),(-0.000001,0.000007,0.000007),(0.000014,-0.000004,-0.000002),(-0.000001,-0.000004,0.000015),(0.000013,-0.000003,0.000002),(0.000009,-0.000002,0.000001),(0.000003,-0.000003,0.000013),(0.000001,-0.000003,0.000009),(0.000006,-0.000021,0.000003),(-0.000001,0.000033,0.000004),(0.000003,-0.000001,0.000001),(-0.000012,0.000018,-0.000012),(-0.000009,-0.000012,0.000021),(0.000108,0.000012,0.000166),(-0.000008,0.000020,-0.000010),(-0.000008,0.000020,-0.000010),(0.000022,-0.000013,-0.000011),(-0.000016,0.000001,-0.000004),(-0.000017,-0.000021,0.000032),(-0.000017,0.000034,-0.000019),(-0.000002,0.000006,-0.000014),(0.000031,-0.000021,-0.000019),(0.000058,0.000034,0.000070),(0.000011,0.000006,-0.000017),(-0.000008,0.000002,-0.000012),(0.000010,-0.000019,0.000007),(0.000028,0.000035,0.000056),(-0.000016,0.000008,0.000010),(0.000021,-0.000019,-0.000019),(-0.000016,-0.000021,0.000022),(0.000023,-0.000023,-0.000008),(0.000007,-0.000011,-0.000014),(-0.000005,-0.000023,0.000021),(-0.000010,-0.000013,0.000005),(-0.000018,-0.000049,-0.000029),(0.000003,0.000100,0.000020),(-0.000023,-0.000037,-0.000030),(-0.000023,0.000058,-0.000029),(-0.000041,-0.000054,0.000001),(0.000149,0.000116,0.000241),(-0.000040,-0.000004,-0.000046),(-0.000022,0.000066,-0.000011),(0.000008,-0.000054,-0.000048),(-0.000032,0.000016,-0.000039),(-0.000012,-0.000062,-0.000021),(0.000003,-0.000041,-0.000050),(-0.000046,-0.000047,-0.000002),(0.000007,-0.000061,-0.000044),(-0.000016,-0.000018,-0.000045),(-0.000035,-0.000062,-0.000001),(-0.000038,-0.000023,-0.000025),(-0.000040,-0.000078,-0.000057),(0.000006,0.000207,0.000035),(-0.000058,-0.000088,-0.000072),(-0.000028,0.000128,-0.000030),(-0.000083,-0.000109,-0.000022),(0.000146,0.000282,0.000245),(-0.000084,-0.000038,-0.000096),(-0.000027,0.000142,-0.000004),(-0.000007,-0.000109,-0.000095),(-0.000047,0.000050,-0.000053),(-0.000038,-0.000118,-0.000055),(-0.000003,-0.000105,-0.000083),(-0.000026,-0.000015,-0.000070),(-0.000070,-0.000109,-0.000020),(-0.000064,-0.000024,-0.000042),(-0.000057,-0.000111,-0.000077),(-0.000002,0.000337,0.000033),(-0.000100,-0.000144,-0.000121),(-0.000057,0.000094,-0.000062),(-0.000065,-0.000183,-0.000088),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,0.000000),(-0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(0.000000,-0.000000,0.000000),(0.000000,-0.000000,0.000000),(-0.000004,0.000013,0.000007),(0.000001,0.000011,-0.000001),(-0.000000,-0.000009,0.000014)};
    //double dd2[221] = {(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000001,0.000001,-0.000001),(-0.000000,0.000001,-0.000000),(-0.000001,0.000000,-0.000001),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000001),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000017,0.000027,-0.000011),(0.000003,0.000011,0.000005),(0.000010,0.000011,0.000009),(0.000005,0.000000,0.000001),(0.000003,0.000002,0.000007),(-0.000028,0.000011,-0.000050),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(0.000001,0.000010,0.000001),(0.000002,0.000001,0.000002),(-0.000008,0.000008,-0.000008),(0.000003,0.000002,0.000006),(-0.000005,0.000008,-0.000008),(-0.000008,0.000005,-0.000008),(0.000006,0.000001,0.000003),(-0.000008,0.000008,-0.000004),(0.000002,0.000002,0.000007),(-0.000002,0.000006,-0.000006),(-0.000006,0.000002,-0.000006),(0.000007,0.000001,0.000002),(-0.000005,0.000005,-0.000001),(-0.000011,0.000005,-0.000005),(0.000005,0.000005,0.000005),(-0.000005,0.000011,-0.000005),(-0.000005,0.000005,-0.000011),(0.000002,0.000005,0.000001),(0.000001,0.000006,0.000002),(-0.000002,0.000010,-0.000001),(0.000005,0.000007,0.000003),(-0.000001,0.000010,-0.000002),(0.000003,0.000008,0.000004),(0.000011,0.000022,0.000012),(-0.000008,0.000006,-0.000012),(-0.000021,0.000022,-0.000020),(0.000007,-0.000005,0.000015),(-0.000010,0.000022,-0.000020),(-0.000080,0.000147,-0.000094),(-0.000022,0.000011,-0.000021),(0.000013,-0.000005,0.000001),(-0.000019,0.000020,-0.000008),(0.000007,-0.000006,0.000026),(-0.000000,0.000012,-0.000008),(-0.000013,0.000002,-0.000012),(0.000022,-0.000008,0.000003),(-0.000006,0.000008,0.000002),(-0.000056,0.000071,-0.000057),(-0.000029,0.000011,-0.000009),(0.000020,0.000001,0.000020),(-0.000009,0.000028,-0.000008),(-0.000036,0.000043,-0.000048),(-0.000012,0.000012,-0.000029),(0.000016,0.000004,0.000008),(0.000007,0.000008,0.000015),(0.000002,0.000021,0.000003),(0.000022,0.000007,0.000014),(0.000002,0.000023,0.000002),(0.000014,0.000010,0.000022),(0.000034,0.000015,0.000036),(-0.000039,0.000016,-0.000054),(-0.000021,0.000024,-0.000017),(-0.000001,-0.000014,0.000013),(-0.000003,0.000022,-0.000013),(-0.000189,0.000199,-0.000226),(-0.000022,0.000007,-0.000018),(0.000009,-0.000009,-0.000019),(-0.000014,0.000018,0.000003),(0.000035,-0.000013,0.000032),(0.000002,0.000031,0.000005),(0.000045,-0.000013,0.000021),(0.000021,-0.000005,0.000046),(0.000022,0.000016,0.000021),(0.000049,-0.000008,0.000029),(0.000017,0.000021,0.000023),(0.000032,-0.000003,0.000050),(0.000060,0.000018,0.000061),(-0.000067,0.000023,-0.000096),(-0.000019,0.000026,-0.000011),(-0.000014,-0.000012,0.000008),(0.000008,0.000023,-0.000004),(-0.000262,0.000177,-0.000321),(-0.000021,0.000001,-0.000012),(0.000009,-0.000006,-0.000047),(-0.000007,0.000016,0.000017),(0.000050,-0.000013,0.000041),(0.000016,0.000038,0.000021),(0.000046,0.000016,0.000040),(0.000077,-0.000010,0.000040),(0.000036,0.000023,0.000049),(0.000048,-0.000006,0.000079),(0.000085,0.000042,0.000083),(-0.000070,0.000022,-0.000113),(-0.000028,0.000039,-0.000017),(0.000072,0.000003,0.000055),(0.000023,0.000061,0.000029),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(0.000000,0.000000,0.000000),(-0.000000,0.000000,-0.000000),(-0.000000,0.000000,-0.000000),(-0.000011,0.000010,-0.000019),(0.000004,0.000001,-0.000003),(0.000006,0.000014,0.000009)};
    //double energy[221] = {1.542830e-42,0.000000e+00,1.017876e-16,4.577482e-41,2.802597e-45,0.000000e+00,2.620864e-16,4.577482e-41,2.620872e-16,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,2.003857e-43,0.000000e+00,-2.317951e+13,1.926055e-04,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,7.229655e-17,4.577482e-41,7.230719e-17,4.577482e-41,0.000000e+00,0.000000e+00,7.230781e-17,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,1.401298e-45,1.401298e-45,1.401298e-45,1.401298e-45,4.577482e-41,6.532545e-27,4.577482e-41,-3.194302e-38,5.898247e-02,4.237519e-37,9.809089e-45,3.363116e-44,0.000000e+00,2.025024e-08,0.000000e+00,2.848895e-08,1.511689e-09,2.150964e-08,0.000000e+00,1.541544e-08,1.854911e-07,2.257209e-08,0.000000e+00,1.605298e-08,7.123480e-09,2.104863e-07,0.000000e+00,9.799036e-05,1.466930e-04,2.309945e-04,0.000000e+00,1.562906e-04,1.148952e-04,1.203459e-04,0.000000e+00,3.012822e-04,4.234389e-05,2.005827e-04,0.000000e+00,8.251942e-05,1.142085e-04,8.502679e-05,0.000000e+00,3.224969e-04,1.431179e-04,3.734327e-04,0.000000e+00,3.013988e-03,1.226029e-03,2.129179e-02,0.000000e+00,1.464419e-02,5.501955e-03,1.881593e-02,0.000000e+00,1.660532e-02,7.767329e-03,9.980457e-03,0.000000e+00,1.278249e-02,1.585113e-03,1.794737e-02,0.000000e+00,1.581253e-02,5.122344e-03,4.728785e-03,-7.521544e-06,-3.115620e+05,7.524181e+01,2.789771e-02,-2.056223e-05,1.973437e+27,4.870370e+01,3.909907e-02,-2.700995e-05,8.916755e-01,6.818591e+01,3.266954e-02,-3.893143e-06,2.683526e-27,2.092911e-23,8.694591e-26,0.000000e+00,9.707691e-21,4.164478e-27,5.105821e-26,0.000000e+00,5.618195e-22,8.011395e-23,7.939312e-27,0.000000e+00,1.583467e-42,1.401298e-45,1.014504e-16,4.577482e-41,2.802597e-45,0.000000e+00,2.620864e-16,4.577482e-41,2.620872e-16,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,2.003857e-43,0.000000e+00,-3.194302e-38,5.898247e-02,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,6.283977e-17,4.577482e-41,6.284513e-17,4.577482e-41,0.000000e+00,0.000000e+00,6.284652e-17,4.577482e-41,0.000000e+00,0.000000e+00,1.399897e-42,1.401298e-45,1.401298e-45,1.401298e-45,1.401298e-45,4.577482e-41,4.237290e-37,9.809089e-45,-2.317951e+13,1.926055e-04,4.237290e-37,9.809089e-45,3.363116e-44,0.000000e+00,2.088217e-27,0.000000e+00,1.568769e-26,6.419391e-27,3.931566e-23,0.000000e+00,6.554724e-27,3.067580e-27,2.328027e-26,0.000000e+00,1.053539e-26,5.014781e-27,7.749388e-24,0.000000e+00,8.620366e-28,2.289166e-27,5.293036e-27,0.000000e+00,6.232589e-27,5.673471e-27,2.332169e-26,0.000000e+00,1.177772e-08,2.611032e-09,1.260247e-08,0.000000e+00,3.821657e-08,2.177334e-09,7.358555e-09,0.000000e+00,4.254145e-09,1.708416e-09,7.111237e-09,0.000000e+00,5.839961e-09,8.409097e-09,4.951757e-09,0.000000e+00,2.549411e-09,1.483298e-09,1.624575e-09,0.000000e+00,1.740956e-09};
    //double factor[221] = {1.542830e-42,0.000000e+00,1.017876e-16,4.577482e-41,2.802597e-45,0.000000e+00,2.620864e-16,4.577482e-41,2.620872e-16,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,2.003857e-43,0.000000e+00,-2.317951e+13,1.926055e-04,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,7.229655e-17,4.577482e-41,7.230719e-17,4.577482e-41,0.000000e+00,0.000000e+00,7.230781e-17,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,1.401298e-45,1.401298e-45,1.401298e-45,1.401298e-45,4.577482e-41,6.530166e-27,4.577482e-41,-3.194302e-38,5.898247e-02,4.237519e-37,9.809089e-45,3.363116e-44,0.000000e+00,6.836695e-12,1.675647e-11,6.620036e-12,6.265301e-12,6.394112e-12,6.476615e-12,7.319163e-12,3.605303e-11,6.644645e-12,6.467787e-12,6.112196e-12,5.683108e-12,6.599283e-12,6.844061e-12,6.563489e-12,6.017392e-12,6.399866e-12,6.154383e-12,3.026520e-07,3.557238e-07,3.342354e-07,3.519032e-07,3.405898e-07,3.512081e-07,3.150973e-07,3.629471e-07,3.268906e-07,3.745598e-07,3.230706e-07,3.582586e-07,3.422037e-07,3.462203e-07,3.601238e-07,3.121515e-07,3.722795e-07,3.343488e-07,3.569448e-07,3.585053e-07,3.710899e-07,3.638215e-07,3.261631e-07,3.464344e-07,3.334570e-07,4.756184e-05,4.773591e-05,4.515327e-05,4.688610e-05,4.413019e-05,4.858421e-05,4.693521e-05,5.063206e-05,4.833565e-05,4.781871e-05,4.642509e-05,4.634951e-05,5.089617e-05,4.829466e-05,5.122057e-05,3.713917e-05,4.961211e-05,4.833119e-05,3.915227e-05,4.985524e-05,4.954117e-05,4.686260e-04,4.303385e-04,3.447189e-04,4.114783e-04,4.361123e-04,4.445273e-04,4.612404e-04,4.645406e-04,4.760396e-04,4.713000e-04,4.726021e-04,4.226256e-04,4.610711e-04,4.390908e-04,1.583467e-42,1.401298e-45,1.014504e-16,4.577482e-41,2.802597e-45,0.000000e+00,2.620864e-16,4.577482e-41,2.620872e-16,4.577482e-41,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,2.003857e-43,0.000000e+00,-3.194302e-38,5.898247e-02,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,6.283977e-17,4.577482e-41,6.284513e-17,4.577482e-41,0.000000e+00,0.000000e+00,6.284652e-17,4.577482e-41,0.000000e+00,0.000000e+00,1.399897e-42,1.401298e-45,1.401298e-45,1.401298e-45,1.401298e-45,4.577482e-41,4.237290e-37,9.809089e-45,-2.317951e+13,1.926055e-04,4.237290e-37,9.809089e-45,3.363116e-44,0.000000e+00,1.856939e-05,4.965303e-05,7.526137e-05,6.964390e-07,1.641322e-04,3.707860e-06,2.637519e-04,2.127130e-04,2.210860e-05,1.228141e-07,5.726594e-04,4.518460e-04,1.110597e-05,1.670070e-05,5.406669e-04,6.669017e-04,8.500725e-05,6.133204e-04,2.710157e-04,1.765020e-04,3.489698e-05,4.957735e-04,4.268136e-04,2.140554e-22,2.745518e-24,1.099713e-26,1.039016e-25,7.914708e-22,1.839824e-20,9.827770e-23,8.790141e-24,4.658280e-18,2.503744e-16,3.010181e-19,5.753074e-22,6.707894e-21,5.924780e-21,1.547127e-26,1.484105e-24,6.045916e-25,5.328845e-06,2.523798e-05,1.002050e-05};
    //double point_weights[221] = {0.000000e+00,0.000000e+00,6.356219e-06,3.324887e-04,3.143648e-02,3.212402e-01,1.299464e-05,7.277725e-04,0.000000e+00,2.066700e+00,2.379370e-01,5.970362e-02,6.884030e-05,1.351831e-03,7.788771e-02,2.020641e-03,1.322531e-06,3.924060e-06,1.305768e-06,1.925648e-04,1.586370e-02,3.594067e-04,1.449458e-01,7.987326e-06,4.113678e-04,0.000000e+00,1.094970e-01,3.780463e-02,0.000000e+00,5.679121e-06,0.000000e+00,2.230565e-04,1.453141e-02,4.649304e-04,0.000000e+00,8.580421e-02,3.249447e-02,0.000000e+00,1.818959e-05,1.134347e-03,1.042623e-01,4.639464e-02,6.318915e-03,1.360220e+02,1.326793e+02,1.313792e+02,1.313792e+02,1.313792e+02,1.227952e+02,1.227954e+02,1.227835e+02,1.335201e+02,1.335201e+02,1.335200e+02,9.516099e+01,9.764464e+01,9.107944e+01,9.713596e+01,1.196765e+02,1.200841e+02,1.316433e+02,1.316505e+02,1.316438e+02,1.316480e+02,1.316498e+02,1.316524e+02,1.772504e+01,1.728354e+01,1.711418e+01,1.711418e+01,1.711418e+01,1.599609e+01,1.599612e+01,1.599554e+01,1.739307e+01,1.739307e+01,1.739307e+01,1.258319e+01,1.272966e+01,1.273444e+01,1.230952e+01,1.270183e+01,1.563527e+01,1.565721e+01,1.714936e+01,1.706164e+01,1.714969e+01,1.714966e+01,1.714972e+01,1.714966e+01,1.709727e+01,1.714977e+01,3.601541e+00,3.511343e+00,3.476937e+00,3.476937e+00,3.476937e+00,3.212072e+00,3.249791e+00,3.249793e+00,3.174251e+00,3.249757e+00,3.533596e+00,3.504852e+00,3.533596e+00,3.533596e+00,2.576064e+00,2.586982e+00,2.587315e+00,2.553255e+00,2.585052e+00,3.180675e+00,3.182183e+00,3.484143e+00,3.477949e+00,3.484163e+00,3.484166e+00,3.484167e+00,3.484161e+00,3.480554e+00,3.484168e+00,9.663483e-01,8.954918e-01,9.421041e-01,9.202220e-01,9.328728e-01,9.328728e-01,9.153517e-01,9.328728e-01,8.686274e-01,8.719293e-01,8.719295e-01,8.649900e-01,8.719268e-01,9.480746e-01,9.456108e-01,9.480746e-01,9.480746e-01,8.537841e-01,8.539019e-01,9.348111e-01,9.343160e-01,9.348124e-01,9.345296e-01,3.132776e-01,3.003089e-01,3.054141e-01,3.011668e-01,3.024214e-01,2.675158e-01,3.024214e-01,3.006469e-01,3.024214e-01,2.823578e-01,2.826646e-01,2.826646e-01,2.819968e-01,2.826644e-01,2.886273e-01,3.073496e-01,3.071240e-01,3.073496e-01,2.925198e-01,3.073496e-01,2.768208e-01,2.768309e-01,3.030502e-01,3.030071e-01,3.030503e-01,3.030261e-01,1.163402e-01,1.128603e-01,1.134196e-01,1.121797e-01,1.123082e-01,1.077705e-01,1.123082e-01,1.121239e-01,1.123082e-01,1.141165e-01,1.141384e-01,1.028049e-01,1.028059e-01,1.125418e-01,1.125377e-01,1.125418e-01,1.125395e-01,4.780666e-02,4.654431e-02,4.660649e-02,4.613614e-02,4.614981e-02,4.558606e-02,4.614981e-02,4.613001e-02,4.614981e-02,4.689961e-02,4.690185e-02,4.624578e-02,4.624538e-02,4.624578e-02,4.624556e-02,2.122287e-02,2.068298e-02,2.069007e-02,2.082095e-02,2.082120e-02,2.571491e-07,3.053828e-05,3.673155e-03,1.025607e-02,0.000000e+00,0.000000e+00,1.847405e-06,1.768821e-07,0.000000e+00,0.000000e+00,4.100356e-04,8.746948e-10,1.113151e-13,4.551340e-11};
    //double partial_densities[221] = {1.016692e-33,2.333626e-34,8.367814e-34,6.744978e-35,4.493371e-36,4.396106e-37,1.908333e-34,4.848228e-35,7.228556e-34,1.717567e-38,6.815741e-37,2.776831e-36,1.143339e-34,3.467837e-35,2.173183e-36,2.919116e-35,3.249745e-13,1.280894e-13,2.872959e-13,5.841356e-14,1.052731e-14,4.778827e-14,2.419135e-15,1.127769e-13,4.739891e-14,2.618893e-13,3.192557e-15,6.648584e-15,4.077459e-07,2.149901e-07,3.746791e-07,1.251948e-07,3.841547e-08,1.084211e-07,3.515827e-07,1.679403e-08,2.794462e-08,3.479711e-05,2.007635e-05,1.120590e-05,2.301993e-06,3.572237e-06,1.566220e-05,5.111461e-31,5.111677e-31,5.111461e-31,5.111597e-31,5.111461e-31,5.111840e-31,5.111245e-31,5.111542e-31,5.111677e-31,5.111786e-31,5.111677e-31,5.111892e-31,5.111867e-31,5.111594e-31,5.111542e-31,5.111461e-31,5.111461e-31,5.111489e-31,5.111542e-31,5.111299e-31,5.111867e-31,5.111786e-31,5.111299e-31,1.202872e-12,1.203062e-12,1.203132e-12,1.203147e-12,1.203080e-12,1.203036e-12,1.203051e-12,1.202950e-12,1.203056e-12,1.203025e-12,1.203067e-12,1.201019e-12,1.202777e-12,1.202918e-12,1.198146e-12,1.202207e-12,1.202323e-12,1.202721e-12,1.202982e-12,1.201824e-12,1.203016e-12,1.203111e-12,1.203109e-12,1.203064e-12,1.202214e-12,1.203094e-12,1.056859e-07,1.069813e-07,1.068918e-07,1.069731e-07,1.067879e-07,9.925203e-08,1.065169e-07,1.067720e-07,9.567351e-08,1.061269e-07,1.068738e-07,1.002584e-07,1.067610e-07,1.069669e-07,1.020017e-07,1.054180e-07,1.061756e-07,9.844285e-08,1.040931e-07,1.043744e-07,1.052831e-07,1.062094e-07,1.033102e-07,1.064327e-07,1.068430e-07,1.067436e-07,1.064088e-07,1.040096e-07,1.067188e-07,5.449067e-06,2.706551e-06,5.836964e-06,3.695634e-06,5.801161e-06,5.833626e-06,3.451926e-06,5.762271e-06,4.397482e-06,5.673675e-06,5.758306e-06,3.997644e-06,5.560026e-06,5.793905e-06,4.525559e-06,5.752838e-06,5.831018e-06,5.168859e-06,5.356980e-06,5.582546e-06,4.978430e-06,5.640204e-06,5.102124e-06,2.601504e-05,9.496978e-06,3.058765e-05,1.358994e-05,3.008526e-05,2.710820e-05,3.054102e-05,1.249342e-05,2.956473e-05,1.764504e-05,2.846226e-05,2.951743e-05,1.532756e-05,2.717309e-05,1.355043e-05,2.998640e-05,1.852374e-05,2.944374e-05,1.020576e-05,3.050481e-05,2.341930e-05,2.510869e-05,2.741899e-05,2.183935e-05,2.806830e-05,2.283266e-05,5.046140e-05,1.499352e-05,6.507664e-05,2.169488e-05,6.326965e-05,4.041267e-05,6.489926e-05,1.996210e-05,6.149049e-05,3.179689e-05,6.107018e-05,4.349307e-05,4.785749e-05,5.461262e-05,3.951244e-05,5.657595e-05,4.188267e-05,6.643681e-05,1.701268e-05,9.193084e-05,2.527302e-05,8.849833e-05,3.780743e-05,9.155914e-05,2.338526e-05,8.530516e-05,3.873166e-05,8.451808e-05,7.331888e-05,4.973260e-05,7.654892e-05,5.304757e-05,7.487628e-05,1.748799e-05,1.078457e-04,4.236045e-05,9.790303e-05,4.905088e-17,3.745068e-17,4.379298e-19,1.966976e-19,5.755041e-15,2.728153e-14,1.389058e-16,3.278741e-16,4.960657e-15,6.570254e-15,9.327300e-19,1.487219e-05,5.521405e-06,1.366262e-05};

    double partial_densities_cpu[10] = {1.016692e-33,2.333626e-34,8.367814e-34,6.744978e-35,4.493371e-36,4.396106e-37,1.908333e-34,4.848228e-35,7.228556e-34,1.717567e-38};
    double point_weights_cpu[10] = {0.000000e+00,0.000000e+00,6.356219e-06,3.324887e-04,3.143648e-02,3.212402e-01,1.299464e-05,7.277725e-04,0.000000e+00,2.066700e+00};
    //double input[number_of_points];
    //for (int i=0; i<number_of_points; i++) {
    //	input[i]=0.001*i;
    //}

    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemset(partial_density_gpu_accum, 0, size);
    hipMemcpy(point_weights_gpu_in, point_weights_cpu, size, hipMemcpyHostToDevice);
    hipMemcpy(partial_density_gpu_in, partial_densities_cpu, size, hipMemcpyHostToDevice);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    // Allocate the host input vectors
    double *energy_cpu = (double *)malloc(size);
    double *factor_cpu = (double *)malloc(size);

    for (int i=0; i<iterations; i++) {
        /////////////////////////////////
	// LIBXC VERSION
        printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        
	gpu_accumulate_point_for_libxc<double,true,true,false><<<blocksPerGrid,threadsPerBlock>>> (point_weights_gpu_in,
    	    number_of_points, block_height,
	    partial_density_gpu_in, dxyz_gpu_in.data, dd1_gpu_in.data, dd2_gpu_in.data,
	    partial_density_gpu_accum, dxyz_gpu_accum.data, dd1_gpu_accum.data, dd2_gpu_accum.data);
	
	
        // Calculate exc_corr and y2a
        libxc_exchange_correlation_gpu<double, true, true, false> (&libxcProxy,
    	    energy_gpu_in,
	    factor_gpu_in,
	    number_of_points,
	    partial_density_gpu_accum,
	    dxyz_gpu_accum.data,
            dd1_gpu_accum.data,
	    dd2_gpu_accum.data);
	
        // Join the results.
        gpu_accumulate_energy_and_forces_from_libxc<double, true, true, false><<<blocksPerGrid, threadsPerBlock>>> (
    	    energy_gpu_in,
	    factor_gpu_in,
	    point_weights_gpu_in,
	    number_of_points,
	    partial_density_gpu_accum);
	

        // Copy the vectors from gpu to cpu
        err = hipMemcpy(energy_cpu, energy_gpu_in, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
        {
    	    printf("Failed to copy vector energy_gpu_in from device to host!\n");
    	    exit(EXIT_FAILURE);
	}

        err = hipMemcpy(factor_cpu, factor_gpu_in, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
        {
    	printf("Failed to copy vector factor_gpu_in from device to host!\n");
	    exit(EXIT_FAILURE);
        }

	///////////////////////////
	// Print libxc results
        printf("iteration %i\n",i);
	print_accumulate_point_data (NULL, NULL, NULL, energy_cpu,
	    factor_cpu, NULL, NULL, number_of_points);

        hipMemset(energy_gpu_in, 0, size);
	hipMemset(factor_gpu_in, 0, size);
        hipMemset(partial_density_gpu_accum, 0, size);

    }

    ////////////////////////////
    // Free Memory CPU
    free(energy_cpu);
    free(factor_cpu);

    ///////////////////////////
    // Free memory GPU
    hipFree (point_weights_gpu_in);
    hipFree (partial_density_gpu_in);
    hipFree (partial_density_gpu_accum);
    hipFree (energy_gpu_in);
    hipFree (factor_gpu_in);

}


/////////////////////////////////////
//// MAIN

int main(int argc, char **argv)
{
    printf("********************************\n");
    printf("** Accumulate Point GPU test  **\n");
    printf("********************************\n");

    //gpu_accumulate_point_test0001();
    //cpu_accumulate_point_test0001();
    //accumulate_data_for_libxc_test0001();
    //accumulate_data_for_libxc_test0002();
    //accumulate_data_for_libxc_test0003();
    //accumulate_data_for_libxc_test0004();
    //accumulate_data_for_libxc_test0005();
    //accumulate_data_for_libxc_test0006();
    //accumulate_data_for_libxc_test0007();
    accumulate_data_for_libxc_test0008(100);

    printf("*************************\n");
    printf("**      Test End       **\n");
    printf("*************************\n");

    return 0;
}